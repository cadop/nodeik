#include "hip/hip_runtime.h"

#include "../native/builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)


using namespace wp;



extern "C" __global__ void compute_loss_cuda_kernel_forward(launch_bounds_t dim,
	array_t<transform> var_body_q,
	int32 var_body_index,
	array_t<float32> var_loss)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    transform var_0;
    vec3 var_1;
    const vec3 var_2 = {0.44999998807907104, 0.0, 0.5};
    vec3 var_3;
    float32 var_4;
    const int32 var_5 = 0;
    //---------
    // forward
        var_0 = wp::load(var_body_q, var_body_index);
        var_1 = wp::transform_get_translation(var_0);
        var_3 = wp::sub(var_1, var_2);
        var_4 = wp::dot(var_3, var_3);
        wp::store(var_loss, var_5, var_4);

}


extern "C" __global__ void compute_loss_cuda_kernel_backward(launch_bounds_t dim,
	array_t<transform> var_body_q,
	int32 var_body_index,
	array_t<float32> var_loss,
	array_t<transform> adj_body_q,
	int32 adj_body_index,
	array_t<float32> adj_loss)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    transform var_0;
    vec3 var_1;
    const vec3 var_2 = {0.44999998807907104, 0.0, 0.5};
    vec3 var_3;
    float32 var_4;
    const int32 var_5 = 0;
    //---------
    // dual vars
    transform adj_0 = 0;
    vec3 adj_1 = 0;
    vec3 adj_2 = 0;
    vec3 adj_3 = 0;
    float32 adj_4 = 0;
    int32 adj_5 = 0;
        //---------
        // forward
        var_0 = wp::load(var_body_q, var_body_index);
        var_1 = wp::transform_get_translation(var_0);
        var_3 = wp::sub(var_1, var_2);
        var_4 = wp::dot(var_3, var_3);
        //wp::store(var_loss, var_5, var_4);
        //---------
        // reverse
        wp::adj_store(var_loss, var_5, var_4, adj_loss, adj_5, adj_4);
        wp::adj_dot(var_3, var_3, adj_3, adj_3, adj_4);
        wp::adj_sub(var_1, var_2, adj_1, adj_2, adj_3);
        wp::adj_transform_get_translation(var_0, adj_0, adj_1);
        wp::adj_load(var_body_q, var_body_index, adj_body_q, adj_body_index, adj_0);
        return;

}



extern "C" {

// Python entry points
WP_API void compute_loss_cuda_forward(void* stream, launch_bounds_t dim,
	array_t<transform> var_body_q,
	int32 var_body_index,
	array_t<float32> var_loss)
{
    compute_loss_cuda_kernel_forward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_body_q,
			var_body_index,
			var_loss);
}

WP_API void compute_loss_cuda_backward(void* stream, launch_bounds_t dim,
	array_t<transform> var_body_q,
	int32 var_body_index,
	array_t<float32> var_loss,
	array_t<transform> adj_body_q,
	int32 adj_body_index,
	array_t<float32> adj_loss)
{
    compute_loss_cuda_kernel_backward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_body_q,
			var_body_index,
			var_loss,
			adj_body_q,
			adj_body_index,
			adj_loss);
}

} // extern C



extern "C" __global__ void step_kernel_cuda_kernel_forward(launch_bounds_t dim,
	array_t<float32> var_x,
	array_t<float32> var_grad,
	float32 var_alpha)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    int32 var_0;
    float32 var_1;
    float32 var_2;
    float32 var_3;
    float32 var_4;
    //---------
    // forward
        var_0 = wp::tid();
        var_1 = wp::load(var_x, var_0);
        var_2 = wp::load(var_grad, var_0);
        var_3 = wp::mul(var_2, var_alpha);
        var_4 = wp::sub(var_1, var_3);
        wp::store(var_x, var_0, var_4);

}


extern "C" __global__ void step_kernel_cuda_kernel_backward(launch_bounds_t dim,
	array_t<float32> var_x,
	array_t<float32> var_grad,
	float32 var_alpha,
	array_t<float32> adj_x,
	array_t<float32> adj_grad,
	float32 adj_alpha)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    int32 var_0;
    float32 var_1;
    float32 var_2;
    float32 var_3;
    float32 var_4;
    //---------
    // dual vars
    int32 adj_0 = 0;
    float32 adj_1 = 0;
    float32 adj_2 = 0;
    float32 adj_3 = 0;
    float32 adj_4 = 0;
        //---------
        // forward
        var_0 = wp::tid();
        var_1 = wp::load(var_x, var_0);
        var_2 = wp::load(var_grad, var_0);
        var_3 = wp::mul(var_2, var_alpha);
        var_4 = wp::sub(var_1, var_3);
        //wp::store(var_x, var_0, var_4);
        //---------
        // reverse
        wp::adj_store(var_x, var_0, var_4, adj_x, adj_0, adj_4);
        wp::adj_sub(var_1, var_3, adj_1, adj_3, adj_4);
        wp::adj_mul(var_2, var_alpha, adj_2, adj_alpha, adj_3);
        wp::adj_load(var_grad, var_0, adj_grad, adj_0, adj_2);
        wp::adj_load(var_x, var_0, adj_x, adj_0, adj_1);
        return;

}



extern "C" {

// Python entry points
WP_API void step_kernel_cuda_forward(void* stream, launch_bounds_t dim,
	array_t<float32> var_x,
	array_t<float32> var_grad,
	float32 var_alpha)
{
    step_kernel_cuda_kernel_forward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_x,
			var_grad,
			var_alpha);
}

WP_API void step_kernel_cuda_backward(void* stream, launch_bounds_t dim,
	array_t<float32> var_x,
	array_t<float32> var_grad,
	float32 var_alpha,
	array_t<float32> adj_x,
	array_t<float32> adj_grad,
	float32 adj_alpha)
{
    step_kernel_cuda_kernel_backward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_x,
			var_grad,
			var_alpha,
			adj_x,
			adj_grad,
			adj_alpha);
}

} // extern C

