#include "hip/hip_runtime.h"

#include "../native/builtin.h"

// avoid namespacing of float type for casting to float type, this is to avoid wp::float(x), which is not valid in C++
#define float(x) cast_float(x)
#define adj_float(x, adj_x, adj_ret) adj_cast_float(x, adj_x, adj_ret)

#define int(x) cast_int(x)
#define adj_int(x, adj_x, adj_ret) adj_cast_int(x, adj_x, adj_ret)


using namespace wp;


static CUDA_CALLABLE vec3 quat_decompose(quat var_q)
{
        //---------
    // primal vars
    const float32 var_0 = 1.0;
    const float32 var_1 = 0.0;
    vec3 var_2;
    vec3 var_3;
    vec3 var_4;
    vec3 var_5;
    vec3 var_6;
    vec3 var_7;
    mat33 var_8;
    const int32 var_9 = 1;
    const int32 var_10 = 2;
    float32 var_11;
    float32 var_12;
    float32 var_13;
    const int32 var_14 = 0;
    float32 var_15;
    float32 var_16;
    float32 var_17;
    float32 var_18;
    float32 var_19;
    float32 var_20;
    vec3 var_21;
    vec3 var_22;
    //---------
    // forward
    var_2 = wp::vec3(var_0, var_1, var_1);
    var_3 = wp::quat_rotate(var_q, var_2);
    var_4 = wp::vec3(var_1, var_0, var_1);
    var_5 = wp::quat_rotate(var_q, var_4);
    var_6 = wp::vec3(var_1, var_1, var_0);
    var_7 = wp::quat_rotate(var_q, var_6);
    var_8 = wp::mat33(var_3, var_5, var_7);
    var_11 = wp::index(var_8, var_9, var_10);
    var_12 = wp::index(var_8, var_10, var_10);
    var_13 = wp::atan2(var_11, var_12);
    var_15 = wp::index(var_8, var_14, var_10);
    var_16 = wp::neg(var_15);
    var_17 = wp::asin(var_16);
    var_18 = wp::index(var_8, var_14, var_9);
    var_19 = wp::index(var_8, var_14, var_14);
    var_20 = wp::atan2(var_18, var_19);
    var_21 = wp::vec3(var_13, var_17, var_20);
    var_22 = wp::neg(var_21);
    return var_22;

}

static CUDA_CALLABLE void adj_quat_decompose(quat var_q,
	quat & adj_q,
	vec3 & adj_ret)
{
        //---------
    // primal vars
    const float32 var_0 = 1.0;
    const float32 var_1 = 0.0;
    vec3 var_2;
    vec3 var_3;
    vec3 var_4;
    vec3 var_5;
    vec3 var_6;
    vec3 var_7;
    mat33 var_8;
    const int32 var_9 = 1;
    const int32 var_10 = 2;
    float32 var_11;
    float32 var_12;
    float32 var_13;
    const int32 var_14 = 0;
    float32 var_15;
    float32 var_16;
    float32 var_17;
    float32 var_18;
    float32 var_19;
    float32 var_20;
    vec3 var_21;
    vec3 var_22;
    //---------
    // dual vars
    float32 adj_0 = 0;
    float32 adj_1 = 0;
    vec3 adj_2 = 0;
    vec3 adj_3 = 0;
    vec3 adj_4 = 0;
    vec3 adj_5 = 0;
    vec3 adj_6 = 0;
    vec3 adj_7 = 0;
    mat33 adj_8 = 0;
    int32 adj_9 = 0;
    int32 adj_10 = 0;
    float32 adj_11 = 0;
    float32 adj_12 = 0;
    float32 adj_13 = 0;
    int32 adj_14 = 0;
    float32 adj_15 = 0;
    float32 adj_16 = 0;
    float32 adj_17 = 0;
    float32 adj_18 = 0;
    float32 adj_19 = 0;
    float32 adj_20 = 0;
    vec3 adj_21 = 0;
    vec3 adj_22 = 0;
    //---------
    // forward
    var_2 = wp::vec3(var_0, var_1, var_1);
    var_3 = wp::quat_rotate(var_q, var_2);
    var_4 = wp::vec3(var_1, var_0, var_1);
    var_5 = wp::quat_rotate(var_q, var_4);
    var_6 = wp::vec3(var_1, var_1, var_0);
    var_7 = wp::quat_rotate(var_q, var_6);
    var_8 = wp::mat33(var_3, var_5, var_7);
    var_11 = wp::index(var_8, var_9, var_10);
    var_12 = wp::index(var_8, var_10, var_10);
    var_13 = wp::atan2(var_11, var_12);
    var_15 = wp::index(var_8, var_14, var_10);
    var_16 = wp::neg(var_15);
    var_17 = wp::asin(var_16);
    var_18 = wp::index(var_8, var_14, var_9);
    var_19 = wp::index(var_8, var_14, var_14);
    var_20 = wp::atan2(var_18, var_19);
    var_21 = wp::vec3(var_13, var_17, var_20);
    var_22 = wp::neg(var_21);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_22 += adj_ret;
    wp::adj_neg(var_21, adj_21, adj_22);
    wp::adj_vec3(var_13, var_17, var_20, adj_13, adj_17, adj_20, adj_21);
    wp::adj_atan2(var_18, var_19, adj_18, adj_19, adj_20);
    wp::adj_index(var_8, var_14, var_14, adj_8, adj_14, adj_14, adj_19);
    wp::adj_index(var_8, var_14, var_9, adj_8, adj_14, adj_9, adj_18);
    wp::adj_asin(var_16, adj_16, adj_17);
    wp::adj_neg(var_15, adj_15, adj_16);
    wp::adj_index(var_8, var_14, var_10, adj_8, adj_14, adj_10, adj_15);
    wp::adj_atan2(var_11, var_12, adj_11, adj_12, adj_13);
    wp::adj_index(var_8, var_10, var_10, adj_8, adj_10, adj_10, adj_12);
    wp::adj_index(var_8, var_9, var_10, adj_8, adj_9, adj_10, adj_11);
    wp::adj_mat33(var_3, var_5, var_7, adj_3, adj_5, adj_7, adj_8);
    wp::adj_quat_rotate(var_q, var_6, adj_q, adj_6, adj_7);
    wp::adj_vec3(var_1, var_1, var_0, adj_1, adj_1, adj_0, adj_6);
    wp::adj_quat_rotate(var_q, var_4, adj_q, adj_4, adj_5);
    wp::adj_vec3(var_1, var_0, var_1, adj_1, adj_0, adj_1, adj_4);
    wp::adj_quat_rotate(var_q, var_2, adj_q, adj_2, adj_3);
    wp::adj_vec3(var_0, var_1, var_1, adj_0, adj_1, adj_1, adj_2);
    return;

}


static CUDA_CALLABLE quat quat_twist(vec3 var_axis,
	quat var_q)
{
        //---------
    // primal vars
    const int32 var_0 = 0;
    float32 var_1;
    const int32 var_2 = 1;
    float32 var_3;
    const int32 var_4 = 2;
    float32 var_5;
    vec3 var_6;
    float32 var_7;
    vec3 var_8;
    float32 var_9;
    float32 var_10;
    float32 var_11;
    const int32 var_12 = 3;
    float32 var_13;
    quat var_14;
    quat var_15;
    //---------
    // forward
    var_1 = wp::index(var_q, var_0);
    var_3 = wp::index(var_q, var_2);
    var_5 = wp::index(var_q, var_4);
    var_6 = wp::vec3(var_1, var_3, var_5);
    var_7 = wp::dot(var_6, var_axis);
    var_8 = wp::mul(var_7, var_axis);
    var_9 = wp::index(var_8, var_0);
    var_10 = wp::index(var_8, var_2);
    var_11 = wp::index(var_8, var_4);
    var_13 = wp::index(var_q, var_12);
    var_14 = wp::quat(var_9, var_10, var_11, var_13);
    var_15 = wp::normalize(var_14);
    return var_15;

}

static CUDA_CALLABLE void adj_quat_twist(vec3 var_axis,
	quat var_q,
	vec3 & adj_axis,
	quat & adj_q,
	quat & adj_ret)
{
        //---------
    // primal vars
    const int32 var_0 = 0;
    float32 var_1;
    const int32 var_2 = 1;
    float32 var_3;
    const int32 var_4 = 2;
    float32 var_5;
    vec3 var_6;
    float32 var_7;
    vec3 var_8;
    float32 var_9;
    float32 var_10;
    float32 var_11;
    const int32 var_12 = 3;
    float32 var_13;
    quat var_14;
    quat var_15;
    //---------
    // dual vars
    int32 adj_0 = 0;
    float32 adj_1 = 0;
    int32 adj_2 = 0;
    float32 adj_3 = 0;
    int32 adj_4 = 0;
    float32 adj_5 = 0;
    vec3 adj_6 = 0;
    float32 adj_7 = 0;
    vec3 adj_8 = 0;
    float32 adj_9 = 0;
    float32 adj_10 = 0;
    float32 adj_11 = 0;
    int32 adj_12 = 0;
    float32 adj_13 = 0;
    quat adj_14 = 0;
    quat adj_15 = 0;
    //---------
    // forward
    var_1 = wp::index(var_q, var_0);
    var_3 = wp::index(var_q, var_2);
    var_5 = wp::index(var_q, var_4);
    var_6 = wp::vec3(var_1, var_3, var_5);
    var_7 = wp::dot(var_6, var_axis);
    var_8 = wp::mul(var_7, var_axis);
    var_9 = wp::index(var_8, var_0);
    var_10 = wp::index(var_8, var_2);
    var_11 = wp::index(var_8, var_4);
    var_13 = wp::index(var_q, var_12);
    var_14 = wp::quat(var_9, var_10, var_11, var_13);
    var_15 = wp::normalize(var_14);
    goto label0;
    //---------
    // reverse
    label0:;
    adj_15 += adj_ret;
    wp::adj_normalize(var_14, adj_14, adj_15);
    wp::adj_quat(var_9, var_10, var_11, var_13, adj_9, adj_10, adj_11, adj_13, adj_14);
    wp::adj_index(var_q, var_12, adj_q, adj_12, adj_13);
    wp::adj_index(var_8, var_4, adj_8, adj_4, adj_11);
    wp::adj_index(var_8, var_2, adj_8, adj_2, adj_10);
    wp::adj_index(var_8, var_0, adj_8, adj_0, adj_9);
    wp::adj_mul(var_7, var_axis, adj_7, adj_axis, adj_8);
    wp::adj_dot(var_6, var_axis, adj_6, adj_axis, adj_7);
    wp::adj_vec3(var_1, var_3, var_5, adj_1, adj_3, adj_5, adj_6);
    wp::adj_index(var_q, var_4, adj_q, adj_4, adj_5);
    wp::adj_index(var_q, var_2, adj_q, adj_2, adj_3);
    wp::adj_index(var_q, var_0, adj_q, adj_0, adj_1);
    return;

}



extern "C" __global__ void eval_articulation_fk_cuda_kernel_forward(launch_bounds_t dim,
	array_t<int32> var_articulation_start,
	array_t<int32> var_articulation_mask,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<vec3> var_body_com,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    int32 var_0;
    int32 var_1;
    const int32 var_2 = 0;
    bool var_3;
    int32 var_4;
    const int32 var_5 = 1;
    int32 var_6;
    int32 var_7;
    range_t var_8;
    int32 var_9;
    int32 var_10;
    transform var_11;
    spatial_vector var_12;
    bool var_13;
    transform var_14;
    spatial_vector var_15;
    transform var_16;
    spatial_vector var_17;
    int32 var_18;
    vec3 var_19;
    transform var_20;
    transform var_21;
    int32 var_22;
    int32 var_23;
    const int32 var_24 = 0;
    bool var_25;
    float32 var_26;
    float32 var_27;
    vec3 var_28;
    quat var_29;
    transform var_30;
    vec3 var_31;
    vec3 var_32;
    spatial_vector var_33;
    const int32 var_34 = 1;
    bool var_35;
    float32 var_36;
    float32 var_37;
    vec3 var_38;
    quat var_39;
    transform var_40;
    vec3 var_41;
    vec3 var_42;
    spatial_vector var_43;
    float32 var_44;
    float32 var_45;
    transform var_46;
    spatial_vector var_47;
    const int32 var_48 = 2;
    bool var_49;
    int32 var_50;
    float32 var_51;
    int32 var_52;
    float32 var_53;
    const int32 var_54 = 2;
    int32 var_55;
    float32 var_56;
    const int32 var_57 = 3;
    int32 var_58;
    float32 var_59;
    quat var_60;
    int32 var_61;
    float32 var_62;
    int32 var_63;
    float32 var_64;
    int32 var_65;
    float32 var_66;
    vec3 var_67;
    vec3 var_68;
    transform var_69;
    vec3 var_70;
    spatial_vector var_71;
    transform var_72;
    spatial_vector var_73;
    const int32 var_74 = 3;
    bool var_75;
    transform var_76;
    vec3 var_77;
    vec3 var_78;
    spatial_vector var_79;
    transform var_80;
    spatial_vector var_81;
    const int32 var_82 = 4;
    bool var_83;
    int32 var_84;
    float32 var_85;
    int32 var_86;
    float32 var_87;
    int32 var_88;
    float32 var_89;
    vec3 var_90;
    int32 var_91;
    float32 var_92;
    const int32 var_93 = 4;
    int32 var_94;
    float32 var_95;
    const int32 var_96 = 5;
    int32 var_97;
    float32 var_98;
    const int32 var_99 = 6;
    int32 var_100;
    float32 var_101;
    quat var_102;
    transform var_103;
    int32 var_104;
    float32 var_105;
    int32 var_106;
    float32 var_107;
    int32 var_108;
    float32 var_109;
    vec3 var_110;
    int32 var_111;
    float32 var_112;
    int32 var_113;
    float32 var_114;
    int32 var_115;
    float32 var_116;
    vec3 var_117;
    spatial_vector var_118;
    transform var_119;
    spatial_vector var_120;
    transform var_121;
    spatial_vector var_122;
    const int32 var_123 = 5;
    bool var_124;
    quat var_125;
    const float32 var_126 = 1.0;
    const float32 var_127 = 0.0;
    vec3 var_128;
    vec3 var_129;
    vec3 var_130;
    vec3 var_131;
    vec3 var_132;
    vec3 var_133;
    vec3 var_134;
    int32 var_135;
    float32 var_136;
    quat var_137;
    vec3 var_138;
    int32 var_139;
    float32 var_140;
    quat var_141;
    quat var_142;
    vec3 var_143;
    int32 var_144;
    float32 var_145;
    quat var_146;
    vec3 var_147;
    quat var_148;
    quat var_149;
    transform var_150;
    int32 var_151;
    float32 var_152;
    vec3 var_153;
    int32 var_154;
    float32 var_155;
    vec3 var_156;
    vec3 var_157;
    int32 var_158;
    float32 var_159;
    vec3 var_160;
    vec3 var_161;
    vec3 var_162;
    spatial_vector var_163;
    transform var_164;
    spatial_vector var_165;
    transform var_166;
    spatial_vector var_167;
    transform var_168;
    spatial_vector var_169;
    const int32 var_170 = 6;
    bool var_171;
    quat var_172;
    vec3 var_173;
    vec3 var_174;
    vec3 var_175;
    vec3 var_176;
    vec3 var_177;
    int32 var_178;
    float32 var_179;
    quat var_180;
    vec3 var_181;
    int32 var_182;
    float32 var_183;
    quat var_184;
    vec3 var_185;
    quat var_186;
    transform var_187;
    int32 var_188;
    float32 var_189;
    vec3 var_190;
    int32 var_191;
    float32 var_192;
    vec3 var_193;
    vec3 var_194;
    vec3 var_195;
    spatial_vector var_196;
    transform var_197;
    spatial_vector var_198;
    transform var_199;
    spatial_vector var_200;
    transform var_201;
    spatial_vector var_202;
    quat var_203;
    vec3 var_204;
    vec3 var_205;
    vec3 var_206;
    quat var_207;
    vec3 var_208;
    quat var_209;
    transform var_210;
    transform var_211;
    vec3 var_212;
    vec3 var_213;
    vec3 var_214;
    vec3 var_215;
    vec3 var_216;
    vec3 var_217;
    vec3 var_218;
    spatial_vector var_219;
    spatial_vector var_220;
    //---------
    // forward
        var_0 = wp::tid();
        if (var_articulation_mask) {
        	var_1 = wp::load(var_articulation_mask, var_0);
        	var_3 = (var_1 == var_2);
        	if (var_3) {
        		return;
        	}
        }
        var_4 = wp::load(var_articulation_start, var_0);
        var_6 = wp::add(var_0, var_5);
        var_7 = wp::load(var_articulation_start, var_6);
        var_8 = wp::range(var_4, var_7);
        for_start_1:;
        	if (iter_cmp(var_8) == 0) goto for_end_1;
        	var_9 = wp::iter_next(var_8);
        	var_10 = wp::load(var_joint_parent, var_9);
        	var_11 = wp::transform_identity();
        	var_12 = wp::spatial_vector();
        	var_13 = (var_10 >= var_2);
        	if (var_13) {
        		var_14 = wp::load(var_body_q, var_10);
        		var_15 = wp::load(var_body_qd, var_10);
        	}
        	var_16 = wp::select(var_13, var_11, var_14);
        	var_17 = wp::select(var_13, var_12, var_15);
        	var_18 = wp::load(var_joint_type, var_9);
        	var_19 = wp::load(var_joint_axis, var_9);
        	var_20 = wp::load(var_joint_X_p, var_9);
        	var_21 = wp::load(var_joint_X_c, var_9);
        	var_22 = wp::load(var_joint_q_start, var_9);
        	var_23 = wp::load(var_joint_qd_start, var_9);
        	var_25 = (var_18 == var_24);
        	if (var_25) {
        		var_26 = wp::load(var_joint_q, var_22);
        		var_27 = wp::load(var_joint_qd, var_23);
        		var_28 = wp::mul(var_19, var_26);
        		var_29 = wp::quat_identity();
        		var_30 = wp::transform(var_28, var_29);
        		var_31 = wp::vec3();
        		var_32 = wp::mul(var_19, var_27);
        		var_33 = wp::spatial_vector(var_31, var_32);
        	}
        	var_35 = (var_18 == var_34);
        	if (var_35) {
        		var_36 = wp::load(var_joint_q, var_22);
        		var_37 = wp::load(var_joint_qd, var_23);
        		var_38 = wp::vec3();
        		var_39 = wp::quat_from_axis_angle(var_19, var_36);
        		var_40 = wp::transform(var_38, var_39);
        		var_41 = wp::mul(var_19, var_37);
        		var_42 = wp::vec3();
        		var_43 = wp::spatial_vector(var_41, var_42);
        	}
        	var_44 = wp::select(var_35, var_26, var_36);
        	var_45 = wp::select(var_35, var_27, var_37);
        	var_46 = wp::select(var_35, var_30, var_40);
        	var_47 = wp::select(var_35, var_33, var_43);
        	var_49 = (var_18 == var_48);
        	if (var_49) {
        		var_50 = wp::add(var_22, var_2);
        		var_51 = wp::load(var_joint_q, var_50);
        		var_52 = wp::add(var_22, var_5);
        		var_53 = wp::load(var_joint_q, var_52);
        		var_55 = wp::add(var_22, var_54);
        		var_56 = wp::load(var_joint_q, var_55);
        		var_58 = wp::add(var_22, var_57);
        		var_59 = wp::load(var_joint_q, var_58);
        		var_60 = wp::quat(var_51, var_53, var_56, var_59);
        		var_61 = wp::add(var_23, var_2);
        		var_62 = wp::load(var_joint_qd, var_61);
        		var_63 = wp::add(var_23, var_5);
        		var_64 = wp::load(var_joint_qd, var_63);
        		var_65 = wp::add(var_23, var_54);
        		var_66 = wp::load(var_joint_qd, var_65);
        		var_67 = wp::vec3(var_62, var_64, var_66);
        		var_68 = wp::vec3();
        		var_69 = wp::transform(var_68, var_60);
        		var_70 = wp::vec3();
        		var_71 = wp::spatial_vector(var_67, var_70);
        	}
        	var_72 = wp::select(var_49, var_46, var_69);
        	var_73 = wp::select(var_49, var_47, var_71);
        	var_75 = (var_18 == var_74);
        	if (var_75) {
        		var_76 = wp::transform_identity();
        		var_77 = wp::vec3();
        		var_78 = wp::vec3();
        		var_79 = wp::spatial_vector(var_77, var_78);
        	}
        	var_80 = wp::select(var_75, var_72, var_76);
        	var_81 = wp::select(var_75, var_73, var_79);
        	var_83 = (var_18 == var_82);
        	if (var_83) {
        		var_84 = wp::add(var_22, var_2);
        		var_85 = wp::load(var_joint_q, var_84);
        		var_86 = wp::add(var_22, var_5);
        		var_87 = wp::load(var_joint_q, var_86);
        		var_88 = wp::add(var_22, var_54);
        		var_89 = wp::load(var_joint_q, var_88);
        		var_90 = wp::vec3(var_85, var_87, var_89);
        		var_91 = wp::add(var_22, var_57);
        		var_92 = wp::load(var_joint_q, var_91);
        		var_94 = wp::add(var_22, var_93);
        		var_95 = wp::load(var_joint_q, var_94);
        		var_97 = wp::add(var_22, var_96);
        		var_98 = wp::load(var_joint_q, var_97);
        		var_100 = wp::add(var_22, var_99);
        		var_101 = wp::load(var_joint_q, var_100);
        		var_102 = wp::quat(var_92, var_95, var_98, var_101);
        		var_103 = wp::transform(var_90, var_102);
        		var_104 = wp::add(var_23, var_2);
        		var_105 = wp::load(var_joint_qd, var_104);
        		var_106 = wp::add(var_23, var_5);
        		var_107 = wp::load(var_joint_qd, var_106);
        		var_108 = wp::add(var_23, var_54);
        		var_109 = wp::load(var_joint_qd, var_108);
        		var_110 = wp::vec3(var_105, var_107, var_109);
        		var_111 = wp::add(var_23, var_57);
        		var_112 = wp::load(var_joint_qd, var_111);
        		var_113 = wp::add(var_23, var_93);
        		var_114 = wp::load(var_joint_qd, var_113);
        		var_115 = wp::add(var_23, var_96);
        		var_116 = wp::load(var_joint_qd, var_115);
        		var_117 = wp::vec3(var_112, var_114, var_116);
        		var_118 = wp::spatial_vector(var_110, var_117);
        		wp::copy(var_119, var_103);
        		wp::copy(var_120, var_118);
        	}
        	var_121 = wp::select(var_83, var_80, var_119);
        	var_122 = wp::select(var_83, var_81, var_120);
        	var_124 = (var_18 == var_123);
        	if (var_124) {
        		var_125 = wp::transform_get_rotation(var_21);
        		var_128 = wp::vec3(var_126, var_127, var_127);
        		var_129 = wp::quat_rotate(var_125, var_128);
        		var_130 = wp::vec3(var_127, var_126, var_127);
        		var_131 = wp::quat_rotate(var_125, var_130);
        		var_132 = wp::vec3(var_127, var_127, var_126);
        		var_133 = wp::quat_rotate(var_125, var_132);
        		wp::copy(var_134, var_129);
        		var_135 = wp::add(var_22, var_2);
        		var_136 = wp::load(var_joint_q, var_135);
        		var_137 = wp::quat_from_axis_angle(var_134, var_136);
        		var_138 = wp::quat_rotate(var_137, var_131);
        		var_139 = wp::add(var_22, var_5);
        		var_140 = wp::load(var_joint_q, var_139);
        		var_141 = wp::quat_from_axis_angle(var_138, var_140);
        		var_142 = wp::mul(var_141, var_137);
        		var_143 = wp::quat_rotate(var_142, var_133);
        		var_144 = wp::add(var_22, var_54);
        		var_145 = wp::load(var_joint_q, var_144);
        		var_146 = wp::quat_from_axis_angle(var_143, var_145);
        		var_147 = wp::vec3();
        		var_148 = wp::mul(var_146, var_141);
        		var_149 = wp::mul(var_148, var_137);
        		var_150 = wp::transform(var_147, var_149);
        		var_151 = wp::add(var_23, var_2);
        		var_152 = wp::load(var_joint_qd, var_151);
        		var_153 = wp::mul(var_134, var_152);
        		var_154 = wp::add(var_23, var_5);
        		var_155 = wp::load(var_joint_qd, var_154);
        		var_156 = wp::mul(var_138, var_155);
        		var_157 = wp::add(var_153, var_156);
        		var_158 = wp::add(var_23, var_54);
        		var_159 = wp::load(var_joint_qd, var_158);
        		var_160 = wp::mul(var_143, var_159);
        		var_161 = wp::add(var_157, var_160);
        		var_162 = wp::vec3();
        		var_163 = wp::spatial_vector(var_161, var_162);
        		wp::copy(var_164, var_150);
        		wp::copy(var_165, var_163);
        	}
        	var_166 = wp::select(var_124, var_121, var_164);
        	var_167 = wp::select(var_124, var_122, var_165);
        	var_168 = wp::select(var_124, var_103, var_150);
        	var_169 = wp::select(var_124, var_118, var_163);
        	var_171 = (var_18 == var_170);
        	if (var_171) {
        		var_172 = wp::transform_get_rotation(var_21);
        		var_173 = wp::vec3(var_126, var_127, var_127);
        		var_174 = wp::quat_rotate(var_172, var_173);
        		var_175 = wp::vec3(var_127, var_126, var_127);
        		var_176 = wp::quat_rotate(var_172, var_175);
        		wp::copy(var_177, var_174);
        		var_178 = wp::add(var_22, var_2);
        		var_179 = wp::load(var_joint_q, var_178);
        		var_180 = wp::quat_from_axis_angle(var_177, var_179);
        		var_181 = wp::quat_rotate(var_180, var_176);
        		var_182 = wp::add(var_22, var_5);
        		var_183 = wp::load(var_joint_q, var_182);
        		var_184 = wp::quat_from_axis_angle(var_181, var_183);
        		var_185 = wp::vec3();
        		var_186 = wp::mul(var_184, var_180);
        		var_187 = wp::transform(var_185, var_186);
        		var_188 = wp::add(var_23, var_2);
        		var_189 = wp::load(var_joint_qd, var_188);
        		var_190 = wp::mul(var_177, var_189);
        		var_191 = wp::add(var_23, var_5);
        		var_192 = wp::load(var_joint_qd, var_191);
        		var_193 = wp::mul(var_181, var_192);
        		var_194 = wp::add(var_190, var_193);
        		var_195 = wp::vec3();
        		var_196 = wp::spatial_vector(var_194, var_195);
        		wp::copy(var_197, var_187);
        		wp::copy(var_198, var_196);
        	}
        	var_199 = wp::select(var_171, var_166, var_197);
        	var_200 = wp::select(var_171, var_167, var_198);
        	var_201 = wp::select(var_171, var_168, var_187);
        	var_202 = wp::select(var_171, var_169, var_196);
        	var_203 = wp::select(var_171, var_125, var_172);
        	var_204 = wp::select(var_171, var_129, var_174);
        	var_205 = wp::select(var_171, var_131, var_176);
        	var_206 = wp::select(var_171, var_134, var_177);
        	var_207 = wp::select(var_171, var_137, var_180);
        	var_208 = wp::select(var_171, var_138, var_181);
        	var_209 = wp::select(var_171, var_141, var_184);
        	var_210 = wp::mul(var_16, var_20);
        	var_211 = wp::mul(var_210, var_199);
        	var_212 = wp::spatial_top(var_200);
        	var_213 = wp::transform_vector(var_210, var_212);
        	var_214 = wp::spatial_bottom(var_200);
        	var_215 = wp::transform_vector(var_210, var_214);
        	var_216 = wp::load(var_body_com, var_9);
        	var_217 = wp::cross(var_213, var_216);
        	var_218 = wp::add(var_215, var_217);
        	var_219 = wp::spatial_vector(var_213, var_218);
        	var_220 = wp::add(var_17, var_219);
        	wp::store(var_body_q, var_9, var_211);
        	wp::store(var_body_qd, var_9, var_220);
        	goto for_start_1;
        for_end_1:;

}


extern "C" __global__ void eval_articulation_fk_cuda_kernel_backward(launch_bounds_t dim,
	array_t<int32> var_articulation_start,
	array_t<int32> var_articulation_mask,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<vec3> var_body_com,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd,
	array_t<int32> adj_articulation_start,
	array_t<int32> adj_articulation_mask,
	array_t<float32> adj_joint_q,
	array_t<float32> adj_joint_qd,
	array_t<int32> adj_joint_q_start,
	array_t<int32> adj_joint_qd_start,
	array_t<int32> adj_joint_type,
	array_t<int32> adj_joint_parent,
	array_t<transform> adj_joint_X_p,
	array_t<transform> adj_joint_X_c,
	array_t<vec3> adj_joint_axis,
	array_t<vec3> adj_body_com,
	array_t<transform> adj_body_q,
	array_t<spatial_vector> adj_body_qd)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    int32 var_0;
    int32 var_1;
    const int32 var_2 = 0;
    bool var_3;
    int32 var_4;
    const int32 var_5 = 1;
    int32 var_6;
    int32 var_7;
    range_t var_8;
    int32 var_9;
    int32 var_10;
    transform var_11;
    spatial_vector var_12;
    bool var_13;
    transform var_14;
    spatial_vector var_15;
    transform var_16;
    spatial_vector var_17;
    int32 var_18;
    vec3 var_19;
    transform var_20;
    transform var_21;
    int32 var_22;
    int32 var_23;
    const int32 var_24 = 0;
    bool var_25;
    float32 var_26;
    float32 var_27;
    vec3 var_28;
    quat var_29;
    transform var_30;
    vec3 var_31;
    vec3 var_32;
    spatial_vector var_33;
    const int32 var_34 = 1;
    bool var_35;
    float32 var_36;
    float32 var_37;
    vec3 var_38;
    quat var_39;
    transform var_40;
    vec3 var_41;
    vec3 var_42;
    spatial_vector var_43;
    float32 var_44;
    float32 var_45;
    transform var_46;
    spatial_vector var_47;
    const int32 var_48 = 2;
    bool var_49;
    int32 var_50;
    float32 var_51;
    int32 var_52;
    float32 var_53;
    const int32 var_54 = 2;
    int32 var_55;
    float32 var_56;
    const int32 var_57 = 3;
    int32 var_58;
    float32 var_59;
    quat var_60;
    int32 var_61;
    float32 var_62;
    int32 var_63;
    float32 var_64;
    int32 var_65;
    float32 var_66;
    vec3 var_67;
    vec3 var_68;
    transform var_69;
    vec3 var_70;
    spatial_vector var_71;
    transform var_72;
    spatial_vector var_73;
    const int32 var_74 = 3;
    bool var_75;
    transform var_76;
    vec3 var_77;
    vec3 var_78;
    spatial_vector var_79;
    transform var_80;
    spatial_vector var_81;
    const int32 var_82 = 4;
    bool var_83;
    int32 var_84;
    float32 var_85;
    int32 var_86;
    float32 var_87;
    int32 var_88;
    float32 var_89;
    vec3 var_90;
    int32 var_91;
    float32 var_92;
    const int32 var_93 = 4;
    int32 var_94;
    float32 var_95;
    const int32 var_96 = 5;
    int32 var_97;
    float32 var_98;
    const int32 var_99 = 6;
    int32 var_100;
    float32 var_101;
    quat var_102;
    transform var_103;
    int32 var_104;
    float32 var_105;
    int32 var_106;
    float32 var_107;
    int32 var_108;
    float32 var_109;
    vec3 var_110;
    int32 var_111;
    float32 var_112;
    int32 var_113;
    float32 var_114;
    int32 var_115;
    float32 var_116;
    vec3 var_117;
    spatial_vector var_118;
    transform var_119;
    spatial_vector var_120;
    transform var_121;
    spatial_vector var_122;
    const int32 var_123 = 5;
    bool var_124;
    quat var_125;
    const float32 var_126 = 1.0;
    const float32 var_127 = 0.0;
    vec3 var_128;
    vec3 var_129;
    vec3 var_130;
    vec3 var_131;
    vec3 var_132;
    vec3 var_133;
    vec3 var_134;
    int32 var_135;
    float32 var_136;
    quat var_137;
    vec3 var_138;
    int32 var_139;
    float32 var_140;
    quat var_141;
    quat var_142;
    vec3 var_143;
    int32 var_144;
    float32 var_145;
    quat var_146;
    vec3 var_147;
    quat var_148;
    quat var_149;
    transform var_150;
    int32 var_151;
    float32 var_152;
    vec3 var_153;
    int32 var_154;
    float32 var_155;
    vec3 var_156;
    vec3 var_157;
    int32 var_158;
    float32 var_159;
    vec3 var_160;
    vec3 var_161;
    vec3 var_162;
    spatial_vector var_163;
    transform var_164;
    spatial_vector var_165;
    transform var_166;
    spatial_vector var_167;
    transform var_168;
    spatial_vector var_169;
    const int32 var_170 = 6;
    bool var_171;
    quat var_172;
    vec3 var_173;
    vec3 var_174;
    vec3 var_175;
    vec3 var_176;
    vec3 var_177;
    int32 var_178;
    float32 var_179;
    quat var_180;
    vec3 var_181;
    int32 var_182;
    float32 var_183;
    quat var_184;
    vec3 var_185;
    quat var_186;
    transform var_187;
    int32 var_188;
    float32 var_189;
    vec3 var_190;
    int32 var_191;
    float32 var_192;
    vec3 var_193;
    vec3 var_194;
    vec3 var_195;
    spatial_vector var_196;
    transform var_197;
    spatial_vector var_198;
    transform var_199;
    spatial_vector var_200;
    transform var_201;
    spatial_vector var_202;
    quat var_203;
    vec3 var_204;
    vec3 var_205;
    vec3 var_206;
    quat var_207;
    vec3 var_208;
    quat var_209;
    transform var_210;
    transform var_211;
    vec3 var_212;
    vec3 var_213;
    vec3 var_214;
    vec3 var_215;
    vec3 var_216;
    vec3 var_217;
    vec3 var_218;
    spatial_vector var_219;
    spatial_vector var_220;
    //---------
    // dual vars
    int32 adj_0 = 0;
    int32 adj_1 = 0;
    int32 adj_2 = 0;
    bool adj_3 = 0;
    int32 adj_4 = 0;
    int32 adj_5 = 0;
    int32 adj_6 = 0;
    int32 adj_7 = 0;
    range_t adj_8 = 0;
    int32 adj_9 = 0;
    int32 adj_10 = 0;
    transform adj_11 = 0;
    spatial_vector adj_12 = 0;
    bool adj_13 = 0;
    transform adj_14 = 0;
    spatial_vector adj_15 = 0;
    transform adj_16 = 0;
    spatial_vector adj_17 = 0;
    int32 adj_18 = 0;
    vec3 adj_19 = 0;
    transform adj_20 = 0;
    transform adj_21 = 0;
    int32 adj_22 = 0;
    int32 adj_23 = 0;
    int32 adj_24 = 0;
    bool adj_25 = 0;
    float32 adj_26 = 0;
    float32 adj_27 = 0;
    vec3 adj_28 = 0;
    quat adj_29 = 0;
    transform adj_30 = 0;
    vec3 adj_31 = 0;
    vec3 adj_32 = 0;
    spatial_vector adj_33 = 0;
    int32 adj_34 = 0;
    bool adj_35 = 0;
    float32 adj_36 = 0;
    float32 adj_37 = 0;
    vec3 adj_38 = 0;
    quat adj_39 = 0;
    transform adj_40 = 0;
    vec3 adj_41 = 0;
    vec3 adj_42 = 0;
    spatial_vector adj_43 = 0;
    float32 adj_44 = 0;
    float32 adj_45 = 0;
    transform adj_46 = 0;
    spatial_vector adj_47 = 0;
    int32 adj_48 = 0;
    bool adj_49 = 0;
    int32 adj_50 = 0;
    float32 adj_51 = 0;
    int32 adj_52 = 0;
    float32 adj_53 = 0;
    int32 adj_54 = 0;
    int32 adj_55 = 0;
    float32 adj_56 = 0;
    int32 adj_57 = 0;
    int32 adj_58 = 0;
    float32 adj_59 = 0;
    quat adj_60 = 0;
    int32 adj_61 = 0;
    float32 adj_62 = 0;
    int32 adj_63 = 0;
    float32 adj_64 = 0;
    int32 adj_65 = 0;
    float32 adj_66 = 0;
    vec3 adj_67 = 0;
    vec3 adj_68 = 0;
    transform adj_69 = 0;
    vec3 adj_70 = 0;
    spatial_vector adj_71 = 0;
    transform adj_72 = 0;
    spatial_vector adj_73 = 0;
    int32 adj_74 = 0;
    bool adj_75 = 0;
    transform adj_76 = 0;
    vec3 adj_77 = 0;
    vec3 adj_78 = 0;
    spatial_vector adj_79 = 0;
    transform adj_80 = 0;
    spatial_vector adj_81 = 0;
    int32 adj_82 = 0;
    bool adj_83 = 0;
    int32 adj_84 = 0;
    float32 adj_85 = 0;
    int32 adj_86 = 0;
    float32 adj_87 = 0;
    int32 adj_88 = 0;
    float32 adj_89 = 0;
    vec3 adj_90 = 0;
    int32 adj_91 = 0;
    float32 adj_92 = 0;
    int32 adj_93 = 0;
    int32 adj_94 = 0;
    float32 adj_95 = 0;
    int32 adj_96 = 0;
    int32 adj_97 = 0;
    float32 adj_98 = 0;
    int32 adj_99 = 0;
    int32 adj_100 = 0;
    float32 adj_101 = 0;
    quat adj_102 = 0;
    transform adj_103 = 0;
    int32 adj_104 = 0;
    float32 adj_105 = 0;
    int32 adj_106 = 0;
    float32 adj_107 = 0;
    int32 adj_108 = 0;
    float32 adj_109 = 0;
    vec3 adj_110 = 0;
    int32 adj_111 = 0;
    float32 adj_112 = 0;
    int32 adj_113 = 0;
    float32 adj_114 = 0;
    int32 adj_115 = 0;
    float32 adj_116 = 0;
    vec3 adj_117 = 0;
    spatial_vector adj_118 = 0;
    transform adj_119 = 0;
    spatial_vector adj_120 = 0;
    transform adj_121 = 0;
    spatial_vector adj_122 = 0;
    int32 adj_123 = 0;
    bool adj_124 = 0;
    quat adj_125 = 0;
    float32 adj_126 = 0;
    float32 adj_127 = 0;
    vec3 adj_128 = 0;
    vec3 adj_129 = 0;
    vec3 adj_130 = 0;
    vec3 adj_131 = 0;
    vec3 adj_132 = 0;
    vec3 adj_133 = 0;
    vec3 adj_134 = 0;
    int32 adj_135 = 0;
    float32 adj_136 = 0;
    quat adj_137 = 0;
    vec3 adj_138 = 0;
    int32 adj_139 = 0;
    float32 adj_140 = 0;
    quat adj_141 = 0;
    quat adj_142 = 0;
    vec3 adj_143 = 0;
    int32 adj_144 = 0;
    float32 adj_145 = 0;
    quat adj_146 = 0;
    vec3 adj_147 = 0;
    quat adj_148 = 0;
    quat adj_149 = 0;
    transform adj_150 = 0;
    int32 adj_151 = 0;
    float32 adj_152 = 0;
    vec3 adj_153 = 0;
    int32 adj_154 = 0;
    float32 adj_155 = 0;
    vec3 adj_156 = 0;
    vec3 adj_157 = 0;
    int32 adj_158 = 0;
    float32 adj_159 = 0;
    vec3 adj_160 = 0;
    vec3 adj_161 = 0;
    vec3 adj_162 = 0;
    spatial_vector adj_163 = 0;
    transform adj_164 = 0;
    spatial_vector adj_165 = 0;
    transform adj_166 = 0;
    spatial_vector adj_167 = 0;
    transform adj_168 = 0;
    spatial_vector adj_169 = 0;
    int32 adj_170 = 0;
    bool adj_171 = 0;
    quat adj_172 = 0;
    vec3 adj_173 = 0;
    vec3 adj_174 = 0;
    vec3 adj_175 = 0;
    vec3 adj_176 = 0;
    vec3 adj_177 = 0;
    int32 adj_178 = 0;
    float32 adj_179 = 0;
    quat adj_180 = 0;
    vec3 adj_181 = 0;
    int32 adj_182 = 0;
    float32 adj_183 = 0;
    quat adj_184 = 0;
    vec3 adj_185 = 0;
    quat adj_186 = 0;
    transform adj_187 = 0;
    int32 adj_188 = 0;
    float32 adj_189 = 0;
    vec3 adj_190 = 0;
    int32 adj_191 = 0;
    float32 adj_192 = 0;
    vec3 adj_193 = 0;
    vec3 adj_194 = 0;
    vec3 adj_195 = 0;
    spatial_vector adj_196 = 0;
    transform adj_197 = 0;
    spatial_vector adj_198 = 0;
    transform adj_199 = 0;
    spatial_vector adj_200 = 0;
    transform adj_201 = 0;
    spatial_vector adj_202 = 0;
    quat adj_203 = 0;
    vec3 adj_204 = 0;
    vec3 adj_205 = 0;
    vec3 adj_206 = 0;
    quat adj_207 = 0;
    vec3 adj_208 = 0;
    quat adj_209 = 0;
    transform adj_210 = 0;
    transform adj_211 = 0;
    vec3 adj_212 = 0;
    vec3 adj_213 = 0;
    vec3 adj_214 = 0;
    vec3 adj_215 = 0;
    vec3 adj_216 = 0;
    vec3 adj_217 = 0;
    vec3 adj_218 = 0;
    spatial_vector adj_219 = 0;
    spatial_vector adj_220 = 0;
        //---------
        // forward
        var_0 = wp::tid();
        if (var_articulation_mask) {
        	var_1 = wp::load(var_articulation_mask, var_0);
        	var_3 = (var_1 == var_2);
        	if (var_3) {
        		goto label0;
        	}
        }
        var_4 = wp::load(var_articulation_start, var_0);
        var_6 = wp::add(var_0, var_5);
        var_7 = wp::load(var_articulation_start, var_6);
        var_8 = wp::range(var_4, var_7);
        //---------
        // reverse
        var_8 = wp::iter_reverse(var_8);
        for_start_1:;
        	if (iter_cmp(var_8) == 0) goto for_end_1;
        	var_9 = wp::iter_next(var_8);
        	adj_10 = 0;
        	adj_11 = 0;
        	adj_12 = 0;
        	adj_13 = 0;
        	adj_14 = 0;
        	adj_15 = 0;
        	adj_16 = 0;
        	adj_17 = 0;
        	adj_18 = 0;
        	adj_19 = 0;
        	adj_20 = 0;
        	adj_21 = 0;
        	adj_22 = 0;
        	adj_23 = 0;
        	adj_24 = 0;
        	adj_25 = 0;
        	adj_26 = 0;
        	adj_27 = 0;
        	adj_28 = 0;
        	adj_29 = 0;
        	adj_30 = 0;
        	adj_31 = 0;
        	adj_32 = 0;
        	adj_33 = 0;
        	adj_34 = 0;
        	adj_35 = 0;
        	adj_36 = 0;
        	adj_37 = 0;
        	adj_38 = 0;
        	adj_39 = 0;
        	adj_40 = 0;
        	adj_41 = 0;
        	adj_42 = 0;
        	adj_43 = 0;
        	adj_44 = 0;
        	adj_45 = 0;
        	adj_46 = 0;
        	adj_47 = 0;
        	adj_48 = 0;
        	adj_49 = 0;
        	adj_50 = 0;
        	adj_51 = 0;
        	adj_52 = 0;
        	adj_53 = 0;
        	adj_54 = 0;
        	adj_55 = 0;
        	adj_56 = 0;
        	adj_57 = 0;
        	adj_58 = 0;
        	adj_59 = 0;
        	adj_60 = 0;
        	adj_61 = 0;
        	adj_62 = 0;
        	adj_63 = 0;
        	adj_64 = 0;
        	adj_65 = 0;
        	adj_66 = 0;
        	adj_67 = 0;
        	adj_68 = 0;
        	adj_69 = 0;
        	adj_70 = 0;
        	adj_71 = 0;
        	adj_72 = 0;
        	adj_73 = 0;
        	adj_74 = 0;
        	adj_75 = 0;
        	adj_76 = 0;
        	adj_77 = 0;
        	adj_78 = 0;
        	adj_79 = 0;
        	adj_80 = 0;
        	adj_81 = 0;
        	adj_82 = 0;
        	adj_83 = 0;
        	adj_84 = 0;
        	adj_85 = 0;
        	adj_86 = 0;
        	adj_87 = 0;
        	adj_88 = 0;
        	adj_89 = 0;
        	adj_90 = 0;
        	adj_91 = 0;
        	adj_92 = 0;
        	adj_93 = 0;
        	adj_94 = 0;
        	adj_95 = 0;
        	adj_96 = 0;
        	adj_97 = 0;
        	adj_98 = 0;
        	adj_99 = 0;
        	adj_100 = 0;
        	adj_101 = 0;
        	adj_102 = 0;
        	adj_103 = 0;
        	adj_104 = 0;
        	adj_105 = 0;
        	adj_106 = 0;
        	adj_107 = 0;
        	adj_108 = 0;
        	adj_109 = 0;
        	adj_110 = 0;
        	adj_111 = 0;
        	adj_112 = 0;
        	adj_113 = 0;
        	adj_114 = 0;
        	adj_115 = 0;
        	adj_116 = 0;
        	adj_117 = 0;
        	adj_118 = 0;
        	adj_119 = 0;
        	adj_120 = 0;
        	adj_121 = 0;
        	adj_122 = 0;
        	adj_123 = 0;
        	adj_124 = 0;
        	adj_125 = 0;
        	adj_126 = 0;
        	adj_127 = 0;
        	adj_128 = 0;
        	adj_129 = 0;
        	adj_130 = 0;
        	adj_131 = 0;
        	adj_132 = 0;
        	adj_133 = 0;
        	adj_134 = 0;
        	adj_135 = 0;
        	adj_136 = 0;
        	adj_137 = 0;
        	adj_138 = 0;
        	adj_139 = 0;
        	adj_140 = 0;
        	adj_141 = 0;
        	adj_142 = 0;
        	adj_143 = 0;
        	adj_144 = 0;
        	adj_145 = 0;
        	adj_146 = 0;
        	adj_147 = 0;
        	adj_148 = 0;
        	adj_149 = 0;
        	adj_150 = 0;
        	adj_151 = 0;
        	adj_152 = 0;
        	adj_153 = 0;
        	adj_154 = 0;
        	adj_155 = 0;
        	adj_156 = 0;
        	adj_157 = 0;
        	adj_158 = 0;
        	adj_159 = 0;
        	adj_160 = 0;
        	adj_161 = 0;
        	adj_162 = 0;
        	adj_163 = 0;
        	adj_164 = 0;
        	adj_165 = 0;
        	adj_166 = 0;
        	adj_167 = 0;
        	adj_168 = 0;
        	adj_169 = 0;
        	adj_170 = 0;
        	adj_171 = 0;
        	adj_172 = 0;
        	adj_173 = 0;
        	adj_174 = 0;
        	adj_175 = 0;
        	adj_176 = 0;
        	adj_177 = 0;
        	adj_178 = 0;
        	adj_179 = 0;
        	adj_180 = 0;
        	adj_181 = 0;
        	adj_182 = 0;
        	adj_183 = 0;
        	adj_184 = 0;
        	adj_185 = 0;
        	adj_186 = 0;
        	adj_187 = 0;
        	adj_188 = 0;
        	adj_189 = 0;
        	adj_190 = 0;
        	adj_191 = 0;
        	adj_192 = 0;
        	adj_193 = 0;
        	adj_194 = 0;
        	adj_195 = 0;
        	adj_196 = 0;
        	adj_197 = 0;
        	adj_198 = 0;
        	adj_199 = 0;
        	adj_200 = 0;
        	adj_201 = 0;
        	adj_202 = 0;
        	adj_203 = 0;
        	adj_204 = 0;
        	adj_205 = 0;
        	adj_206 = 0;
        	adj_207 = 0;
        	adj_208 = 0;
        	adj_209 = 0;
        	adj_210 = 0;
        	adj_211 = 0;
        	adj_212 = 0;
        	adj_213 = 0;
        	adj_214 = 0;
        	adj_215 = 0;
        	adj_216 = 0;
        	adj_217 = 0;
        	adj_218 = 0;
        	adj_219 = 0;
        	adj_220 = 0;
        	var_10 = wp::load(var_joint_parent, var_9);
        	var_11 = wp::transform_identity();
        	var_12 = wp::spatial_vector();
        	var_13 = (var_10 >= var_2);
        	if (var_13) {
        		var_14 = wp::load(var_body_q, var_10);
        		var_15 = wp::load(var_body_qd, var_10);
        	}
        	var_16 = wp::select(var_13, var_11, var_14);
        	var_17 = wp::select(var_13, var_12, var_15);
        	var_18 = wp::load(var_joint_type, var_9);
        	var_19 = wp::load(var_joint_axis, var_9);
        	var_20 = wp::load(var_joint_X_p, var_9);
        	var_21 = wp::load(var_joint_X_c, var_9);
        	var_22 = wp::load(var_joint_q_start, var_9);
        	var_23 = wp::load(var_joint_qd_start, var_9);
        	var_25 = (var_18 == var_24);
        	if (var_25) {
        		var_26 = wp::load(var_joint_q, var_22);
        		var_27 = wp::load(var_joint_qd, var_23);
        		var_28 = wp::mul(var_19, var_26);
        		var_29 = wp::quat_identity();
        		var_30 = wp::transform(var_28, var_29);
        		var_31 = wp::vec3();
        		var_32 = wp::mul(var_19, var_27);
        		var_33 = wp::spatial_vector(var_31, var_32);
        	}
        	var_35 = (var_18 == var_34);
        	if (var_35) {
        		var_36 = wp::load(var_joint_q, var_22);
        		var_37 = wp::load(var_joint_qd, var_23);
        		var_38 = wp::vec3();
        		var_39 = wp::quat_from_axis_angle(var_19, var_36);
        		var_40 = wp::transform(var_38, var_39);
        		var_41 = wp::mul(var_19, var_37);
        		var_42 = wp::vec3();
        		var_43 = wp::spatial_vector(var_41, var_42);
        	}
        	var_44 = wp::select(var_35, var_26, var_36);
        	var_45 = wp::select(var_35, var_27, var_37);
        	var_46 = wp::select(var_35, var_30, var_40);
        	var_47 = wp::select(var_35, var_33, var_43);
        	var_49 = (var_18 == var_48);
        	if (var_49) {
        		var_50 = wp::add(var_22, var_2);
        		var_51 = wp::load(var_joint_q, var_50);
        		var_52 = wp::add(var_22, var_5);
        		var_53 = wp::load(var_joint_q, var_52);
        		var_55 = wp::add(var_22, var_54);
        		var_56 = wp::load(var_joint_q, var_55);
        		var_58 = wp::add(var_22, var_57);
        		var_59 = wp::load(var_joint_q, var_58);
        		var_60 = wp::quat(var_51, var_53, var_56, var_59);
        		var_61 = wp::add(var_23, var_2);
        		var_62 = wp::load(var_joint_qd, var_61);
        		var_63 = wp::add(var_23, var_5);
        		var_64 = wp::load(var_joint_qd, var_63);
        		var_65 = wp::add(var_23, var_54);
        		var_66 = wp::load(var_joint_qd, var_65);
        		var_67 = wp::vec3(var_62, var_64, var_66);
        		var_68 = wp::vec3();
        		var_69 = wp::transform(var_68, var_60);
        		var_70 = wp::vec3();
        		var_71 = wp::spatial_vector(var_67, var_70);
        	}
        	var_72 = wp::select(var_49, var_46, var_69);
        	var_73 = wp::select(var_49, var_47, var_71);
        	var_75 = (var_18 == var_74);
        	if (var_75) {
        		var_76 = wp::transform_identity();
        		var_77 = wp::vec3();
        		var_78 = wp::vec3();
        		var_79 = wp::spatial_vector(var_77, var_78);
        	}
        	var_80 = wp::select(var_75, var_72, var_76);
        	var_81 = wp::select(var_75, var_73, var_79);
        	var_83 = (var_18 == var_82);
        	if (var_83) {
        		var_84 = wp::add(var_22, var_2);
        		var_85 = wp::load(var_joint_q, var_84);
        		var_86 = wp::add(var_22, var_5);
        		var_87 = wp::load(var_joint_q, var_86);
        		var_88 = wp::add(var_22, var_54);
        		var_89 = wp::load(var_joint_q, var_88);
        		var_90 = wp::vec3(var_85, var_87, var_89);
        		var_91 = wp::add(var_22, var_57);
        		var_92 = wp::load(var_joint_q, var_91);
        		var_94 = wp::add(var_22, var_93);
        		var_95 = wp::load(var_joint_q, var_94);
        		var_97 = wp::add(var_22, var_96);
        		var_98 = wp::load(var_joint_q, var_97);
        		var_100 = wp::add(var_22, var_99);
        		var_101 = wp::load(var_joint_q, var_100);
        		var_102 = wp::quat(var_92, var_95, var_98, var_101);
        		var_103 = wp::transform(var_90, var_102);
        		var_104 = wp::add(var_23, var_2);
        		var_105 = wp::load(var_joint_qd, var_104);
        		var_106 = wp::add(var_23, var_5);
        		var_107 = wp::load(var_joint_qd, var_106);
        		var_108 = wp::add(var_23, var_54);
        		var_109 = wp::load(var_joint_qd, var_108);
        		var_110 = wp::vec3(var_105, var_107, var_109);
        		var_111 = wp::add(var_23, var_57);
        		var_112 = wp::load(var_joint_qd, var_111);
        		var_113 = wp::add(var_23, var_93);
        		var_114 = wp::load(var_joint_qd, var_113);
        		var_115 = wp::add(var_23, var_96);
        		var_116 = wp::load(var_joint_qd, var_115);
        		var_117 = wp::vec3(var_112, var_114, var_116);
        		var_118 = wp::spatial_vector(var_110, var_117);
        		wp::copy(var_119, var_103);
        		wp::copy(var_120, var_118);
        	}
        	var_121 = wp::select(var_83, var_80, var_119);
        	var_122 = wp::select(var_83, var_81, var_120);
        	var_124 = (var_18 == var_123);
        	if (var_124) {
        		var_125 = wp::transform_get_rotation(var_21);
        		var_128 = wp::vec3(var_126, var_127, var_127);
        		var_129 = wp::quat_rotate(var_125, var_128);
        		var_130 = wp::vec3(var_127, var_126, var_127);
        		var_131 = wp::quat_rotate(var_125, var_130);
        		var_132 = wp::vec3(var_127, var_127, var_126);
        		var_133 = wp::quat_rotate(var_125, var_132);
        		wp::copy(var_134, var_129);
        		var_135 = wp::add(var_22, var_2);
        		var_136 = wp::load(var_joint_q, var_135);
        		var_137 = wp::quat_from_axis_angle(var_134, var_136);
        		var_138 = wp::quat_rotate(var_137, var_131);
        		var_139 = wp::add(var_22, var_5);
        		var_140 = wp::load(var_joint_q, var_139);
        		var_141 = wp::quat_from_axis_angle(var_138, var_140);
        		var_142 = wp::mul(var_141, var_137);
        		var_143 = wp::quat_rotate(var_142, var_133);
        		var_144 = wp::add(var_22, var_54);
        		var_145 = wp::load(var_joint_q, var_144);
        		var_146 = wp::quat_from_axis_angle(var_143, var_145);
        		var_147 = wp::vec3();
        		var_148 = wp::mul(var_146, var_141);
        		var_149 = wp::mul(var_148, var_137);
        		var_150 = wp::transform(var_147, var_149);
        		var_151 = wp::add(var_23, var_2);
        		var_152 = wp::load(var_joint_qd, var_151);
        		var_153 = wp::mul(var_134, var_152);
        		var_154 = wp::add(var_23, var_5);
        		var_155 = wp::load(var_joint_qd, var_154);
        		var_156 = wp::mul(var_138, var_155);
        		var_157 = wp::add(var_153, var_156);
        		var_158 = wp::add(var_23, var_54);
        		var_159 = wp::load(var_joint_qd, var_158);
        		var_160 = wp::mul(var_143, var_159);
        		var_161 = wp::add(var_157, var_160);
        		var_162 = wp::vec3();
        		var_163 = wp::spatial_vector(var_161, var_162);
        		wp::copy(var_164, var_150);
        		wp::copy(var_165, var_163);
        	}
        	var_166 = wp::select(var_124, var_121, var_164);
        	var_167 = wp::select(var_124, var_122, var_165);
        	var_168 = wp::select(var_124, var_103, var_150);
        	var_169 = wp::select(var_124, var_118, var_163);
        	var_171 = (var_18 == var_170);
        	if (var_171) {
        		var_172 = wp::transform_get_rotation(var_21);
        		var_173 = wp::vec3(var_126, var_127, var_127);
        		var_174 = wp::quat_rotate(var_172, var_173);
        		var_175 = wp::vec3(var_127, var_126, var_127);
        		var_176 = wp::quat_rotate(var_172, var_175);
        		wp::copy(var_177, var_174);
        		var_178 = wp::add(var_22, var_2);
        		var_179 = wp::load(var_joint_q, var_178);
        		var_180 = wp::quat_from_axis_angle(var_177, var_179);
        		var_181 = wp::quat_rotate(var_180, var_176);
        		var_182 = wp::add(var_22, var_5);
        		var_183 = wp::load(var_joint_q, var_182);
        		var_184 = wp::quat_from_axis_angle(var_181, var_183);
        		var_185 = wp::vec3();
        		var_186 = wp::mul(var_184, var_180);
        		var_187 = wp::transform(var_185, var_186);
        		var_188 = wp::add(var_23, var_2);
        		var_189 = wp::load(var_joint_qd, var_188);
        		var_190 = wp::mul(var_177, var_189);
        		var_191 = wp::add(var_23, var_5);
        		var_192 = wp::load(var_joint_qd, var_191);
        		var_193 = wp::mul(var_181, var_192);
        		var_194 = wp::add(var_190, var_193);
        		var_195 = wp::vec3();
        		var_196 = wp::spatial_vector(var_194, var_195);
        		wp::copy(var_197, var_187);
        		wp::copy(var_198, var_196);
        	}
        	var_199 = wp::select(var_171, var_166, var_197);
        	var_200 = wp::select(var_171, var_167, var_198);
        	var_201 = wp::select(var_171, var_168, var_187);
        	var_202 = wp::select(var_171, var_169, var_196);
        	var_203 = wp::select(var_171, var_125, var_172);
        	var_204 = wp::select(var_171, var_129, var_174);
        	var_205 = wp::select(var_171, var_131, var_176);
        	var_206 = wp::select(var_171, var_134, var_177);
        	var_207 = wp::select(var_171, var_137, var_180);
        	var_208 = wp::select(var_171, var_138, var_181);
        	var_209 = wp::select(var_171, var_141, var_184);
        	var_210 = wp::mul(var_16, var_20);
        	var_211 = wp::mul(var_210, var_199);
        	var_212 = wp::spatial_top(var_200);
        	var_213 = wp::transform_vector(var_210, var_212);
        	var_214 = wp::spatial_bottom(var_200);
        	var_215 = wp::transform_vector(var_210, var_214);
        	var_216 = wp::load(var_body_com, var_9);
        	var_217 = wp::cross(var_213, var_216);
        	var_218 = wp::add(var_215, var_217);
        	var_219 = wp::spatial_vector(var_213, var_218);
        	var_220 = wp::add(var_17, var_219);
        	//wp::store(var_body_q, var_9, var_211);
        	//wp::store(var_body_qd, var_9, var_220);
        	wp::adj_store(var_body_qd, var_9, var_220, adj_body_qd, adj_9, adj_220);
        	wp::adj_store(var_body_q, var_9, var_211, adj_body_q, adj_9, adj_211);
        	wp::adj_add(var_17, var_219, adj_17, adj_219, adj_220);
        	wp::adj_spatial_vector(var_213, var_218, adj_213, adj_218, adj_219);
        	wp::adj_add(var_215, var_217, adj_215, adj_217, adj_218);
        	wp::adj_cross(var_213, var_216, adj_213, adj_216, adj_217);
        	wp::adj_load(var_body_com, var_9, adj_body_com, adj_9, adj_216);
        	wp::adj_transform_vector(var_210, var_214, adj_210, adj_214, adj_215);
        	wp::adj_spatial_bottom(var_200, adj_200, adj_214);
        	wp::adj_transform_vector(var_210, var_212, adj_210, adj_212, adj_213);
        	wp::adj_spatial_top(var_200, adj_200, adj_212);
        	wp::adj_mul(var_210, var_199, adj_210, adj_199, adj_211);
        	wp::adj_mul(var_16, var_20, adj_16, adj_20, adj_210);
        	wp::adj_select(var_171, var_141, var_184, adj_171, adj_141, adj_184, adj_209);
        	wp::adj_select(var_171, var_138, var_181, adj_171, adj_138, adj_181, adj_208);
        	wp::adj_select(var_171, var_137, var_180, adj_171, adj_137, adj_180, adj_207);
        	wp::adj_select(var_171, var_134, var_177, adj_171, adj_134, adj_177, adj_206);
        	wp::adj_select(var_171, var_131, var_176, adj_171, adj_131, adj_176, adj_205);
        	wp::adj_select(var_171, var_129, var_174, adj_171, adj_129, adj_174, adj_204);
        	wp::adj_select(var_171, var_125, var_172, adj_171, adj_125, adj_172, adj_203);
        	wp::adj_select(var_171, var_169, var_196, adj_171, adj_169, adj_196, adj_202);
        	wp::adj_select(var_171, var_168, var_187, adj_171, adj_168, adj_187, adj_201);
        	wp::adj_select(var_171, var_167, var_198, adj_171, adj_167, adj_198, adj_200);
        	wp::adj_select(var_171, var_166, var_197, adj_171, adj_166, adj_197, adj_199);
        	if (var_171) {
        		wp::adj_copy(var_198, var_196, adj_198, adj_196);
        		wp::adj_copy(var_197, var_187, adj_197, adj_187);
        		wp::adj_spatial_vector(var_194, var_195, adj_194, adj_195, adj_196);
        		wp::adj_add(var_190, var_193, adj_190, adj_193, adj_194);
        		wp::adj_mul(var_181, var_192, adj_181, adj_192, adj_193);
        		wp::adj_load(var_joint_qd, var_191, adj_joint_qd, adj_191, adj_192);
        		wp::adj_add(var_23, var_5, adj_23, adj_5, adj_191);
        		wp::adj_mul(var_177, var_189, adj_177, adj_189, adj_190);
        		wp::adj_load(var_joint_qd, var_188, adj_joint_qd, adj_188, adj_189);
        		wp::adj_add(var_23, var_2, adj_23, adj_2, adj_188);
        		wp::adj_transform(var_185, var_186, adj_185, adj_186, adj_187);
        		wp::adj_mul(var_184, var_180, adj_184, adj_180, adj_186);
        		wp::adj_quat_from_axis_angle(var_181, var_183, adj_181, adj_183, adj_184);
        		wp::adj_load(var_joint_q, var_182, adj_joint_q, adj_182, adj_183);
        		wp::adj_add(var_22, var_5, adj_22, adj_5, adj_182);
        		wp::adj_quat_rotate(var_180, var_176, adj_180, adj_176, adj_181);
        		wp::adj_quat_from_axis_angle(var_177, var_179, adj_177, adj_179, adj_180);
        		wp::adj_load(var_joint_q, var_178, adj_joint_q, adj_178, adj_179);
        		wp::adj_add(var_22, var_2, adj_22, adj_2, adj_178);
        		wp::adj_copy(var_177, var_174, adj_177, adj_174);
        		wp::adj_quat_rotate(var_172, var_175, adj_172, adj_175, adj_176);
        		wp::adj_vec3(var_127, var_126, var_127, adj_127, adj_126, adj_127, adj_175);
        		wp::adj_quat_rotate(var_172, var_173, adj_172, adj_173, adj_174);
        		wp::adj_vec3(var_126, var_127, var_127, adj_126, adj_127, adj_127, adj_173);
        		wp::adj_transform_get_rotation(var_21, adj_21, adj_172);
        	}
        	wp::adj_select(var_124, var_118, var_163, adj_124, adj_118, adj_163, adj_169);
        	wp::adj_select(var_124, var_103, var_150, adj_124, adj_103, adj_150, adj_168);
        	wp::adj_select(var_124, var_122, var_165, adj_124, adj_122, adj_165, adj_167);
        	wp::adj_select(var_124, var_121, var_164, adj_124, adj_121, adj_164, adj_166);
        	if (var_124) {
        		wp::adj_copy(var_165, var_163, adj_165, adj_163);
        		wp::adj_copy(var_164, var_150, adj_164, adj_150);
        		wp::adj_spatial_vector(var_161, var_162, adj_161, adj_162, adj_163);
        		wp::adj_add(var_157, var_160, adj_157, adj_160, adj_161);
        		wp::adj_mul(var_143, var_159, adj_143, adj_159, adj_160);
        		wp::adj_load(var_joint_qd, var_158, adj_joint_qd, adj_158, adj_159);
        		wp::adj_add(var_23, var_54, adj_23, adj_54, adj_158);
        		wp::adj_add(var_153, var_156, adj_153, adj_156, adj_157);
        		wp::adj_mul(var_138, var_155, adj_138, adj_155, adj_156);
        		wp::adj_load(var_joint_qd, var_154, adj_joint_qd, adj_154, adj_155);
        		wp::adj_add(var_23, var_5, adj_23, adj_5, adj_154);
        		wp::adj_mul(var_134, var_152, adj_134, adj_152, adj_153);
        		wp::adj_load(var_joint_qd, var_151, adj_joint_qd, adj_151, adj_152);
        		wp::adj_add(var_23, var_2, adj_23, adj_2, adj_151);
        		wp::adj_transform(var_147, var_149, adj_147, adj_149, adj_150);
        		wp::adj_mul(var_148, var_137, adj_148, adj_137, adj_149);
        		wp::adj_mul(var_146, var_141, adj_146, adj_141, adj_148);
        		wp::adj_quat_from_axis_angle(var_143, var_145, adj_143, adj_145, adj_146);
        		wp::adj_load(var_joint_q, var_144, adj_joint_q, adj_144, adj_145);
        		wp::adj_add(var_22, var_54, adj_22, adj_54, adj_144);
        		wp::adj_quat_rotate(var_142, var_133, adj_142, adj_133, adj_143);
        		wp::adj_mul(var_141, var_137, adj_141, adj_137, adj_142);
        		wp::adj_quat_from_axis_angle(var_138, var_140, adj_138, adj_140, adj_141);
        		wp::adj_load(var_joint_q, var_139, adj_joint_q, adj_139, adj_140);
        		wp::adj_add(var_22, var_5, adj_22, adj_5, adj_139);
        		wp::adj_quat_rotate(var_137, var_131, adj_137, adj_131, adj_138);
        		wp::adj_quat_from_axis_angle(var_134, var_136, adj_134, adj_136, adj_137);
        		wp::adj_load(var_joint_q, var_135, adj_joint_q, adj_135, adj_136);
        		wp::adj_add(var_22, var_2, adj_22, adj_2, adj_135);
        		wp::adj_copy(var_134, var_129, adj_134, adj_129);
        		wp::adj_quat_rotate(var_125, var_132, adj_125, adj_132, adj_133);
        		wp::adj_vec3(var_127, var_127, var_126, adj_127, adj_127, adj_126, adj_132);
        		wp::adj_quat_rotate(var_125, var_130, adj_125, adj_130, adj_131);
        		wp::adj_vec3(var_127, var_126, var_127, adj_127, adj_126, adj_127, adj_130);
        		wp::adj_quat_rotate(var_125, var_128, adj_125, adj_128, adj_129);
        		wp::adj_vec3(var_126, var_127, var_127, adj_126, adj_127, adj_127, adj_128);
        		wp::adj_transform_get_rotation(var_21, adj_21, adj_125);
        	}
        	wp::adj_select(var_83, var_81, var_120, adj_83, adj_81, adj_120, adj_122);
        	wp::adj_select(var_83, var_80, var_119, adj_83, adj_80, adj_119, adj_121);
        	if (var_83) {
        		wp::adj_copy(var_120, var_118, adj_120, adj_118);
        		wp::adj_copy(var_119, var_103, adj_119, adj_103);
        		wp::adj_spatial_vector(var_110, var_117, adj_110, adj_117, adj_118);
        		wp::adj_vec3(var_112, var_114, var_116, adj_112, adj_114, adj_116, adj_117);
        		wp::adj_load(var_joint_qd, var_115, adj_joint_qd, adj_115, adj_116);
        		wp::adj_add(var_23, var_96, adj_23, adj_96, adj_115);
        		wp::adj_load(var_joint_qd, var_113, adj_joint_qd, adj_113, adj_114);
        		wp::adj_add(var_23, var_93, adj_23, adj_93, adj_113);
        		wp::adj_load(var_joint_qd, var_111, adj_joint_qd, adj_111, adj_112);
        		wp::adj_add(var_23, var_57, adj_23, adj_57, adj_111);
        		wp::adj_vec3(var_105, var_107, var_109, adj_105, adj_107, adj_109, adj_110);
        		wp::adj_load(var_joint_qd, var_108, adj_joint_qd, adj_108, adj_109);
        		wp::adj_add(var_23, var_54, adj_23, adj_54, adj_108);
        		wp::adj_load(var_joint_qd, var_106, adj_joint_qd, adj_106, adj_107);
        		wp::adj_add(var_23, var_5, adj_23, adj_5, adj_106);
        		wp::adj_load(var_joint_qd, var_104, adj_joint_qd, adj_104, adj_105);
        		wp::adj_add(var_23, var_2, adj_23, adj_2, adj_104);
        		wp::adj_transform(var_90, var_102, adj_90, adj_102, adj_103);
        		wp::adj_quat(var_92, var_95, var_98, var_101, adj_92, adj_95, adj_98, adj_101, adj_102);
        		wp::adj_load(var_joint_q, var_100, adj_joint_q, adj_100, adj_101);
        		wp::adj_add(var_22, var_99, adj_22, adj_99, adj_100);
        		wp::adj_load(var_joint_q, var_97, adj_joint_q, adj_97, adj_98);
        		wp::adj_add(var_22, var_96, adj_22, adj_96, adj_97);
        		wp::adj_load(var_joint_q, var_94, adj_joint_q, adj_94, adj_95);
        		wp::adj_add(var_22, var_93, adj_22, adj_93, adj_94);
        		wp::adj_load(var_joint_q, var_91, adj_joint_q, adj_91, adj_92);
        		wp::adj_add(var_22, var_57, adj_22, adj_57, adj_91);
        		wp::adj_vec3(var_85, var_87, var_89, adj_85, adj_87, adj_89, adj_90);
        		wp::adj_load(var_joint_q, var_88, adj_joint_q, adj_88, adj_89);
        		wp::adj_add(var_22, var_54, adj_22, adj_54, adj_88);
        		wp::adj_load(var_joint_q, var_86, adj_joint_q, adj_86, adj_87);
        		wp::adj_add(var_22, var_5, adj_22, adj_5, adj_86);
        		wp::adj_load(var_joint_q, var_84, adj_joint_q, adj_84, adj_85);
        		wp::adj_add(var_22, var_2, adj_22, adj_2, adj_84);
        	}
        	wp::adj_select(var_75, var_73, var_79, adj_75, adj_73, adj_79, adj_81);
        	wp::adj_select(var_75, var_72, var_76, adj_75, adj_72, adj_76, adj_80);
        	if (var_75) {
        		wp::adj_spatial_vector(var_77, var_78, adj_77, adj_78, adj_79);
        	}
        	wp::adj_select(var_49, var_47, var_71, adj_49, adj_47, adj_71, adj_73);
        	wp::adj_select(var_49, var_46, var_69, adj_49, adj_46, adj_69, adj_72);
        	if (var_49) {
        		wp::adj_spatial_vector(var_67, var_70, adj_67, adj_70, adj_71);
        		wp::adj_transform(var_68, var_60, adj_68, adj_60, adj_69);
        		wp::adj_vec3(var_62, var_64, var_66, adj_62, adj_64, adj_66, adj_67);
        		wp::adj_load(var_joint_qd, var_65, adj_joint_qd, adj_65, adj_66);
        		wp::adj_add(var_23, var_54, adj_23, adj_54, adj_65);
        		wp::adj_load(var_joint_qd, var_63, adj_joint_qd, adj_63, adj_64);
        		wp::adj_add(var_23, var_5, adj_23, adj_5, adj_63);
        		wp::adj_load(var_joint_qd, var_61, adj_joint_qd, adj_61, adj_62);
        		wp::adj_add(var_23, var_2, adj_23, adj_2, adj_61);
        		wp::adj_quat(var_51, var_53, var_56, var_59, adj_51, adj_53, adj_56, adj_59, adj_60);
        		wp::adj_load(var_joint_q, var_58, adj_joint_q, adj_58, adj_59);
        		wp::adj_add(var_22, var_57, adj_22, adj_57, adj_58);
        		wp::adj_load(var_joint_q, var_55, adj_joint_q, adj_55, adj_56);
        		wp::adj_add(var_22, var_54, adj_22, adj_54, adj_55);
        		wp::adj_load(var_joint_q, var_52, adj_joint_q, adj_52, adj_53);
        		wp::adj_add(var_22, var_5, adj_22, adj_5, adj_52);
        		wp::adj_load(var_joint_q, var_50, adj_joint_q, adj_50, adj_51);
        		wp::adj_add(var_22, var_2, adj_22, adj_2, adj_50);
        	}
        	wp::adj_select(var_35, var_33, var_43, adj_35, adj_33, adj_43, adj_47);
        	wp::adj_select(var_35, var_30, var_40, adj_35, adj_30, adj_40, adj_46);
        	wp::adj_select(var_35, var_27, var_37, adj_35, adj_27, adj_37, adj_45);
        	wp::adj_select(var_35, var_26, var_36, adj_35, adj_26, adj_36, adj_44);
        	if (var_35) {
        		wp::adj_spatial_vector(var_41, var_42, adj_41, adj_42, adj_43);
        		wp::adj_mul(var_19, var_37, adj_19, adj_37, adj_41);
        		wp::adj_transform(var_38, var_39, adj_38, adj_39, adj_40);
        		wp::adj_quat_from_axis_angle(var_19, var_36, adj_19, adj_36, adj_39);
        		wp::adj_load(var_joint_qd, var_23, adj_joint_qd, adj_23, adj_37);
        		wp::adj_load(var_joint_q, var_22, adj_joint_q, adj_22, adj_36);
        	}
        	if (var_25) {
        		wp::adj_spatial_vector(var_31, var_32, adj_31, adj_32, adj_33);
        		wp::adj_mul(var_19, var_27, adj_19, adj_27, adj_32);
        		wp::adj_transform(var_28, var_29, adj_28, adj_29, adj_30);
        		wp::adj_mul(var_19, var_26, adj_19, adj_26, adj_28);
        		wp::adj_load(var_joint_qd, var_23, adj_joint_qd, adj_23, adj_27);
        		wp::adj_load(var_joint_q, var_22, adj_joint_q, adj_22, adj_26);
        	}
        	wp::adj_load(var_joint_qd_start, var_9, adj_joint_qd_start, adj_9, adj_23);
        	wp::adj_load(var_joint_q_start, var_9, adj_joint_q_start, adj_9, adj_22);
        	wp::adj_load(var_joint_X_c, var_9, adj_joint_X_c, adj_9, adj_21);
        	wp::adj_load(var_joint_X_p, var_9, adj_joint_X_p, adj_9, adj_20);
        	wp::adj_load(var_joint_axis, var_9, adj_joint_axis, adj_9, adj_19);
        	wp::adj_load(var_joint_type, var_9, adj_joint_type, adj_9, adj_18);
        	wp::adj_select(var_13, var_12, var_15, adj_13, adj_12, adj_15, adj_17);
        	wp::adj_select(var_13, var_11, var_14, adj_13, adj_11, adj_14, adj_16);
        	if (var_13) {
        		wp::adj_load(var_body_qd, var_10, adj_body_qd, adj_10, adj_15);
        		wp::adj_load(var_body_q, var_10, adj_body_q, adj_10, adj_14);
        	}
        	wp::adj_load(var_joint_parent, var_9, adj_joint_parent, adj_9, adj_10);
        	goto for_start_1;
        for_end_1:;
        wp::adj_range(var_4, var_7, adj_4, adj_7, adj_8);
        wp::adj_load(var_articulation_start, var_6, adj_articulation_start, adj_6, adj_7);
        wp::adj_add(var_0, var_5, adj_0, adj_5, adj_6);
        wp::adj_load(var_articulation_start, var_0, adj_articulation_start, adj_0, adj_4);
        if (var_articulation_mask) {
        	if (var_3) {
        		label0:;
        	}
        	wp::adj_load(var_articulation_mask, var_0, adj_articulation_mask, adj_0, adj_1);
        }
        return;

}



extern "C" {

// Python entry points
WP_API void eval_articulation_fk_cuda_forward(void* stream, launch_bounds_t dim,
	array_t<int32> var_articulation_start,
	array_t<int32> var_articulation_mask,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<vec3> var_body_com,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd)
{
    eval_articulation_fk_cuda_kernel_forward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_articulation_start,
			var_articulation_mask,
			var_joint_q,
			var_joint_qd,
			var_joint_q_start,
			var_joint_qd_start,
			var_joint_type,
			var_joint_parent,
			var_joint_X_p,
			var_joint_X_c,
			var_joint_axis,
			var_body_com,
			var_body_q,
			var_body_qd);
}

WP_API void eval_articulation_fk_cuda_backward(void* stream, launch_bounds_t dim,
	array_t<int32> var_articulation_start,
	array_t<int32> var_articulation_mask,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<vec3> var_body_com,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd,
	array_t<int32> adj_articulation_start,
	array_t<int32> adj_articulation_mask,
	array_t<float32> adj_joint_q,
	array_t<float32> adj_joint_qd,
	array_t<int32> adj_joint_q_start,
	array_t<int32> adj_joint_qd_start,
	array_t<int32> adj_joint_type,
	array_t<int32> adj_joint_parent,
	array_t<transform> adj_joint_X_p,
	array_t<transform> adj_joint_X_c,
	array_t<vec3> adj_joint_axis,
	array_t<vec3> adj_body_com,
	array_t<transform> adj_body_q,
	array_t<spatial_vector> adj_body_qd)
{
    eval_articulation_fk_cuda_kernel_backward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_articulation_start,
			var_articulation_mask,
			var_joint_q,
			var_joint_qd,
			var_joint_q_start,
			var_joint_qd_start,
			var_joint_type,
			var_joint_parent,
			var_joint_X_p,
			var_joint_X_c,
			var_joint_axis,
			var_body_com,
			var_body_q,
			var_body_qd,
			adj_articulation_start,
			adj_articulation_mask,
			adj_joint_q,
			adj_joint_qd,
			adj_joint_q_start,
			adj_joint_qd_start,
			adj_joint_type,
			adj_joint_parent,
			adj_joint_X_p,
			adj_joint_X_c,
			adj_joint_axis,
			adj_body_com,
			adj_body_q,
			adj_body_qd);
}

} // extern C



extern "C" __global__ void eval_articulation_ik_cuda_kernel_forward(launch_bounds_t dim,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd,
	array_t<vec3> var_body_com,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    int32 var_0;
    int32 var_1;
    int32 var_2;
    transform var_3;
    transform var_4;
    transform var_5;
    vec3 var_6;
    vec3 var_7;
    vec3 var_8;
    const int32 var_9 = 0;
    bool var_10;
    transform var_11;
    transform var_12;
    vec3 var_13;
    transform var_14;
    vec3 var_15;
    vec3 var_16;
    vec3 var_17;
    spatial_vector var_18;
    vec3 var_19;
    vec3 var_20;
    vec3 var_21;
    vec3 var_22;
    transform var_23;
    vec3 var_24;
    vec3 var_25;
    transform var_26;
    vec3 var_27;
    transform var_28;
    vec3 var_29;
    vec3 var_30;
    vec3 var_31;
    spatial_vector var_32;
    vec3 var_33;
    vec3 var_34;
    vec3 var_35;
    vec3 var_36;
    int32 var_37;
    vec3 var_38;
    vec3 var_39;
    vec3 var_40;
    quat var_41;
    quat var_42;
    vec3 var_43;
    vec3 var_44;
    vec3 var_45;
    int32 var_46;
    int32 var_47;
    const int32 var_48 = 0;
    bool var_49;
    vec3 var_50;
    float32 var_51;
    float32 var_52;
    const int32 var_53 = 1;
    bool var_54;
    vec3 var_55;
    vec3 var_56;
    quat var_57;
    quat var_58;
    quat var_59;
    const int32 var_60 = 3;
    float32 var_61;
    float32 var_62;
    const float32 var_63 = 2.0;
    float32 var_64;
    float32 var_65;
    const int32 var_66 = 1;
    float32 var_67;
    const int32 var_68 = 2;
    float32 var_69;
    vec3 var_70;
    float32 var_71;
    float32 var_72;
    float32 var_73;
    float32 var_74;
    vec3 var_75;
    float32 var_76;
    float32 var_77;
    const int32 var_78 = 2;
    bool var_79;
    quat var_80;
    quat var_81;
    float32 var_82;
    int32 var_83;
    float32 var_84;
    int32 var_85;
    float32 var_86;
    int32 var_87;
    float32 var_88;
    int32 var_89;
    float32 var_90;
    int32 var_91;
    float32 var_92;
    int32 var_93;
    float32 var_94;
    int32 var_95;
    quat var_96;
    const int32 var_97 = 3;
    bool var_98;
    const int32 var_99 = 4;
    bool var_100;
    quat var_101;
    quat var_102;
    float32 var_103;
    int32 var_104;
    float32 var_105;
    int32 var_106;
    float32 var_107;
    int32 var_108;
    float32 var_109;
    int32 var_110;
    float32 var_111;
    const int32 var_112 = 4;
    int32 var_113;
    float32 var_114;
    const int32 var_115 = 5;
    int32 var_116;
    float32 var_117;
    const int32 var_118 = 6;
    int32 var_119;
    float32 var_120;
    int32 var_121;
    float32 var_122;
    int32 var_123;
    float32 var_124;
    int32 var_125;
    float32 var_126;
    int32 var_127;
    float32 var_128;
    int32 var_129;
    float32 var_130;
    int32 var_131;
    quat var_132;
    const int32 var_133 = 5;
    bool var_134;
    quat var_135;
    quat var_136;
    quat var_137;
    quat var_138;
    quat var_139;
    quat var_140;
    vec3 var_141;
    const float32 var_142 = 1.0;
    const float32 var_143 = 0.0;
    vec3 var_144;
    float32 var_145;
    quat var_146;
    vec3 var_147;
    vec3 var_148;
    float32 var_149;
    quat var_150;
    quat var_151;
    vec3 var_152;
    vec3 var_153;
    float32 var_154;
    quat var_155;
    quat var_156;
    float32 var_157;
    int32 var_158;
    float32 var_159;
    int32 var_160;
    float32 var_161;
    int32 var_162;
    vec3 var_163;
    float32 var_164;
    int32 var_165;
    vec3 var_166;
    float32 var_167;
    int32 var_168;
    vec3 var_169;
    float32 var_170;
    int32 var_171;
    quat var_172;
    const int32 var_173 = 6;
    bool var_174;
    quat var_175;
    quat var_176;
    quat var_177;
    quat var_178;
    quat var_179;
    quat var_180;
    vec3 var_181;
    vec3 var_182;
    float32 var_183;
    quat var_184;
    vec3 var_185;
    vec3 var_186;
    float32 var_187;
    quat var_188;
    quat var_189;
    float32 var_190;
    int32 var_191;
    float32 var_192;
    int32 var_193;
    vec3 var_194;
    float32 var_195;
    int32 var_196;
    vec3 var_197;
    float32 var_198;
    int32 var_199;
    quat var_200;
    quat var_201;
    vec3 var_202;
    vec3 var_203;
    quat var_204;
    vec3 var_205;
    quat var_206;
    quat var_207;
    //---------
    // forward
        var_0 = wp::tid();
        wp::copy(var_1, var_0);
        var_2 = wp::load(var_joint_parent, var_0);
        var_3 = wp::load(var_joint_X_p, var_0);
        var_4 = wp::load(var_joint_X_c, var_0);
        wp::copy(var_5, var_3);
        var_6 = wp::vec3();
        var_7 = wp::vec3();
        var_8 = wp::vec3();
        var_10 = (var_2 >= var_9);
        if (var_10) {
        	var_11 = wp::load(var_body_q, var_2);
        	var_12 = wp::mul(var_11, var_5);
        	var_13 = wp::transform_get_translation(var_12);
        	var_14 = wp::load(var_body_q, var_2);
        	var_15 = wp::load(var_body_com, var_2);
        	var_16 = wp::transform_point(var_14, var_15);
        	var_17 = wp::sub(var_13, var_16);
        	var_18 = wp::load(var_body_qd, var_2);
        	var_19 = wp::spatial_top(var_18);
        	var_20 = wp::spatial_bottom(var_18);
        	var_21 = wp::cross(var_19, var_17);
        	var_22 = wp::add(var_20, var_21);
        }
        var_23 = wp::select(var_10, var_5, var_12);
        var_24 = wp::select(var_10, var_7, var_19);
        var_25 = wp::select(var_10, var_8, var_22);
        var_26 = wp::load(var_body_q, var_1);
        var_27 = wp::transform_get_translation(var_26);
        var_28 = wp::load(var_body_q, var_1);
        var_29 = wp::load(var_body_com, var_1);
        var_30 = wp::transform_point(var_28, var_29);
        var_31 = wp::sub(var_27, var_30);
        var_32 = wp::load(var_body_qd, var_1);
        var_33 = wp::spatial_top(var_32);
        var_34 = wp::spatial_bottom(var_32);
        var_35 = wp::cross(var_33, var_31);
        var_36 = wp::add(var_34, var_35);
        var_37 = wp::load(var_joint_type, var_0);
        var_38 = wp::load(var_joint_axis, var_0);
        var_39 = wp::transform_get_translation(var_23);
        var_40 = wp::transform_get_translation(var_26);
        var_41 = wp::transform_get_rotation(var_23);
        var_42 = wp::transform_get_rotation(var_26);
        var_43 = wp::sub(var_40, var_39);
        var_44 = wp::sub(var_36, var_25);
        var_45 = wp::sub(var_33, var_24);
        var_46 = wp::load(var_joint_q_start, var_0);
        var_47 = wp::load(var_joint_qd_start, var_0);
        var_49 = (var_37 == var_48);
        if (var_49) {
        	var_50 = wp::transform_vector(var_23, var_38);
        	var_51 = wp::dot(var_43, var_50);
        	var_52 = wp::dot(var_44, var_50);
        	wp::store(var_joint_q, var_46, var_51);
        	wp::store(var_joint_qd, var_47, var_52);
        	return;
        }
        var_54 = (var_37 == var_53);
        if (var_54) {
        	var_55 = wp::transform_vector(var_23, var_38);
        	var_56 = wp::transform_vector(var_26, var_38);
        	var_57 = wp::quat_inverse(var_41);
        	var_58 = wp::mul(var_57, var_42);
        	var_59 = quat_twist(var_38, var_58);
        	var_61 = wp::index(var_59, var_60);
        	var_62 = wp::acos(var_61);
        	var_64 = wp::mul(var_62, var_63);
        	var_65 = wp::index(var_59, var_9);
        	var_67 = wp::index(var_59, var_66);
        	var_69 = wp::index(var_59, var_68);
        	var_70 = wp::vec3(var_65, var_67, var_69);
        	var_71 = wp::dot(var_38, var_70);
        	var_72 = wp::sign(var_71);
        	var_73 = wp::mul(var_64, var_72);
        	var_74 = wp::dot(var_45, var_55);
        	wp::store(var_joint_q, var_46, var_73);
        	wp::store(var_joint_qd, var_47, var_74);
        	return;
        }
        var_75 = wp::select(var_54, var_50, var_55);
        var_76 = wp::select(var_54, var_51, var_73);
        var_77 = wp::select(var_54, var_52, var_74);
        var_79 = (var_37 == var_78);
        if (var_79) {
        	var_80 = wp::quat_inverse(var_41);
        	var_81 = wp::mul(var_80, var_42);
        	var_82 = wp::index(var_81, var_9);
        	var_83 = wp::add(var_46, var_9);
        	wp::store(var_joint_q, var_83, var_82);
        	var_84 = wp::index(var_81, var_66);
        	var_85 = wp::add(var_46, var_66);
        	wp::store(var_joint_q, var_85, var_84);
        	var_86 = wp::index(var_81, var_68);
        	var_87 = wp::add(var_46, var_68);
        	wp::store(var_joint_q, var_87, var_86);
        	var_88 = wp::index(var_81, var_60);
        	var_89 = wp::add(var_46, var_60);
        	wp::store(var_joint_q, var_89, var_88);
        	var_90 = wp::index(var_45, var_9);
        	var_91 = wp::add(var_47, var_9);
        	wp::store(var_joint_qd, var_91, var_90);
        	var_92 = wp::index(var_45, var_66);
        	var_93 = wp::add(var_47, var_66);
        	wp::store(var_joint_qd, var_93, var_92);
        	var_94 = wp::index(var_45, var_68);
        	var_95 = wp::add(var_47, var_68);
        	wp::store(var_joint_qd, var_95, var_94);
        	return;
        }
        var_96 = wp::select(var_79, var_58, var_81);
        var_98 = (var_37 == var_97);
        if (var_98) {
        	return;
        }
        var_100 = (var_37 == var_99);
        if (var_100) {
        	var_101 = wp::quat_inverse(var_41);
        	var_102 = wp::mul(var_101, var_42);
        	var_103 = wp::index(var_43, var_9);
        	var_104 = wp::add(var_46, var_9);
        	wp::store(var_joint_q, var_104, var_103);
        	var_105 = wp::index(var_43, var_66);
        	var_106 = wp::add(var_46, var_66);
        	wp::store(var_joint_q, var_106, var_105);
        	var_107 = wp::index(var_43, var_68);
        	var_108 = wp::add(var_46, var_68);
        	wp::store(var_joint_q, var_108, var_107);
        	var_109 = wp::index(var_102, var_9);
        	var_110 = wp::add(var_46, var_60);
        	wp::store(var_joint_q, var_110, var_109);
        	var_111 = wp::index(var_102, var_66);
        	var_113 = wp::add(var_46, var_112);
        	wp::store(var_joint_q, var_113, var_111);
        	var_114 = wp::index(var_102, var_68);
        	var_116 = wp::add(var_46, var_115);
        	wp::store(var_joint_q, var_116, var_114);
        	var_117 = wp::index(var_102, var_60);
        	var_119 = wp::add(var_46, var_118);
        	wp::store(var_joint_q, var_119, var_117);
        	var_120 = wp::index(var_45, var_9);
        	var_121 = wp::add(var_47, var_9);
        	wp::store(var_joint_qd, var_121, var_120);
        	var_122 = wp::index(var_45, var_66);
        	var_123 = wp::add(var_47, var_66);
        	wp::store(var_joint_qd, var_123, var_122);
        	var_124 = wp::index(var_45, var_68);
        	var_125 = wp::add(var_47, var_68);
        	wp::store(var_joint_qd, var_125, var_124);
        	var_126 = wp::index(var_44, var_9);
        	var_127 = wp::add(var_47, var_60);
        	wp::store(var_joint_qd, var_127, var_126);
        	var_128 = wp::index(var_44, var_66);
        	var_129 = wp::add(var_47, var_112);
        	wp::store(var_joint_qd, var_129, var_128);
        	var_130 = wp::index(var_44, var_68);
        	var_131 = wp::add(var_47, var_115);
        	wp::store(var_joint_qd, var_131, var_130);
        }
        var_132 = wp::select(var_100, var_96, var_102);
        var_134 = (var_37 == var_133);
        if (var_134) {
        	var_135 = wp::transform_get_rotation(var_4);
        	var_136 = wp::quat_inverse(var_135);
        	var_137 = wp::quat_inverse(var_41);
        	var_138 = wp::mul(var_136, var_137);
        	var_139 = wp::mul(var_138, var_42);
        	var_140 = wp::mul(var_139, var_135);
        	var_141 = quat_decompose(var_140);
        	var_144 = wp::vec3(var_142, var_143, var_143);
        	var_145 = wp::index(var_141, var_9);
        	var_146 = wp::quat_from_axis_angle(var_144, var_145);
        	var_147 = wp::vec3(var_143, var_142, var_143);
        	var_148 = wp::quat_rotate(var_146, var_147);
        	var_149 = wp::index(var_141, var_66);
        	var_150 = wp::quat_from_axis_angle(var_148, var_149);
        	var_151 = wp::mul(var_150, var_146);
        	var_152 = wp::vec3(var_143, var_143, var_142);
        	var_153 = wp::quat_rotate(var_151, var_152);
        	var_154 = wp::index(var_141, var_68);
        	var_155 = wp::quat_from_axis_angle(var_153, var_154);
        	var_156 = wp::mul(var_41, var_135);
        	var_157 = wp::index(var_141, var_9);
        	var_158 = wp::add(var_46, var_9);
        	wp::store(var_joint_q, var_158, var_157);
        	var_159 = wp::index(var_141, var_66);
        	var_160 = wp::add(var_46, var_66);
        	wp::store(var_joint_q, var_160, var_159);
        	var_161 = wp::index(var_141, var_68);
        	var_162 = wp::add(var_46, var_68);
        	wp::store(var_joint_q, var_162, var_161);
        	var_163 = wp::quat_rotate(var_156, var_144);
        	var_164 = wp::dot(var_163, var_45);
        	var_165 = wp::add(var_47, var_9);
        	wp::store(var_joint_qd, var_165, var_164);
        	var_166 = wp::quat_rotate(var_156, var_148);
        	var_167 = wp::dot(var_166, var_45);
        	var_168 = wp::add(var_47, var_66);
        	wp::store(var_joint_qd, var_168, var_167);
        	var_169 = wp::quat_rotate(var_156, var_153);
        	var_170 = wp::dot(var_169, var_45);
        	var_171 = wp::add(var_47, var_68);
        	wp::store(var_joint_qd, var_171, var_170);
        	return;
        }
        var_172 = wp::select(var_134, var_132, var_140);
        var_174 = (var_37 == var_173);
        if (var_174) {
        	var_175 = wp::transform_get_rotation(var_4);
        	var_176 = wp::quat_inverse(var_175);
        	var_177 = wp::quat_inverse(var_41);
        	var_178 = wp::mul(var_176, var_177);
        	var_179 = wp::mul(var_178, var_42);
        	var_180 = wp::mul(var_179, var_175);
        	var_181 = quat_decompose(var_180);
        	var_182 = wp::vec3(var_142, var_143, var_143);
        	var_183 = wp::index(var_181, var_9);
        	var_184 = wp::quat_from_axis_angle(var_182, var_183);
        	var_185 = wp::vec3(var_143, var_142, var_143);
        	var_186 = wp::quat_rotate(var_184, var_185);
        	var_187 = wp::index(var_181, var_66);
        	var_188 = wp::quat_from_axis_angle(var_186, var_187);
        	var_189 = wp::mul(var_41, var_175);
        	var_190 = wp::index(var_181, var_9);
        	var_191 = wp::add(var_46, var_9);
        	wp::store(var_joint_q, var_191, var_190);
        	var_192 = wp::index(var_181, var_66);
        	var_193 = wp::add(var_46, var_66);
        	wp::store(var_joint_q, var_193, var_192);
        	var_194 = wp::quat_rotate(var_189, var_182);
        	var_195 = wp::dot(var_194, var_45);
        	var_196 = wp::add(var_47, var_9);
        	wp::store(var_joint_qd, var_196, var_195);
        	var_197 = wp::quat_rotate(var_189, var_186);
        	var_198 = wp::dot(var_197, var_45);
        	var_199 = wp::add(var_47, var_66);
        	wp::store(var_joint_qd, var_199, var_198);
        	return;
        }
        var_200 = wp::select(var_174, var_172, var_180);
        var_201 = wp::select(var_174, var_135, var_175);
        var_202 = wp::select(var_174, var_141, var_181);
        var_203 = wp::select(var_174, var_144, var_182);
        var_204 = wp::select(var_174, var_146, var_184);
        var_205 = wp::select(var_174, var_148, var_186);
        var_206 = wp::select(var_174, var_150, var_188);
        var_207 = wp::select(var_174, var_156, var_189);

}


extern "C" __global__ void eval_articulation_ik_cuda_kernel_backward(launch_bounds_t dim,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd,
	array_t<vec3> var_body_com,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd,
	array_t<transform> adj_body_q,
	array_t<spatial_vector> adj_body_qd,
	array_t<vec3> adj_body_com,
	array_t<int32> adj_joint_type,
	array_t<int32> adj_joint_parent,
	array_t<transform> adj_joint_X_p,
	array_t<transform> adj_joint_X_c,
	array_t<vec3> adj_joint_axis,
	array_t<int32> adj_joint_q_start,
	array_t<int32> adj_joint_qd_start,
	array_t<float32> adj_joint_q,
	array_t<float32> adj_joint_qd)
{
    int _idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (_idx >= dim.size) 
        return;

    set_launch_bounds(dim);

        //---------
    // primal vars
    int32 var_0;
    int32 var_1;
    int32 var_2;
    transform var_3;
    transform var_4;
    transform var_5;
    vec3 var_6;
    vec3 var_7;
    vec3 var_8;
    const int32 var_9 = 0;
    bool var_10;
    transform var_11;
    transform var_12;
    vec3 var_13;
    transform var_14;
    vec3 var_15;
    vec3 var_16;
    vec3 var_17;
    spatial_vector var_18;
    vec3 var_19;
    vec3 var_20;
    vec3 var_21;
    vec3 var_22;
    transform var_23;
    vec3 var_24;
    vec3 var_25;
    transform var_26;
    vec3 var_27;
    transform var_28;
    vec3 var_29;
    vec3 var_30;
    vec3 var_31;
    spatial_vector var_32;
    vec3 var_33;
    vec3 var_34;
    vec3 var_35;
    vec3 var_36;
    int32 var_37;
    vec3 var_38;
    vec3 var_39;
    vec3 var_40;
    quat var_41;
    quat var_42;
    vec3 var_43;
    vec3 var_44;
    vec3 var_45;
    int32 var_46;
    int32 var_47;
    const int32 var_48 = 0;
    bool var_49;
    vec3 var_50;
    float32 var_51;
    float32 var_52;
    const int32 var_53 = 1;
    bool var_54;
    vec3 var_55;
    vec3 var_56;
    quat var_57;
    quat var_58;
    quat var_59;
    const int32 var_60 = 3;
    float32 var_61;
    float32 var_62;
    const float32 var_63 = 2.0;
    float32 var_64;
    float32 var_65;
    const int32 var_66 = 1;
    float32 var_67;
    const int32 var_68 = 2;
    float32 var_69;
    vec3 var_70;
    float32 var_71;
    float32 var_72;
    float32 var_73;
    float32 var_74;
    vec3 var_75;
    float32 var_76;
    float32 var_77;
    const int32 var_78 = 2;
    bool var_79;
    quat var_80;
    quat var_81;
    float32 var_82;
    int32 var_83;
    float32 var_84;
    int32 var_85;
    float32 var_86;
    int32 var_87;
    float32 var_88;
    int32 var_89;
    float32 var_90;
    int32 var_91;
    float32 var_92;
    int32 var_93;
    float32 var_94;
    int32 var_95;
    quat var_96;
    const int32 var_97 = 3;
    bool var_98;
    const int32 var_99 = 4;
    bool var_100;
    quat var_101;
    quat var_102;
    float32 var_103;
    int32 var_104;
    float32 var_105;
    int32 var_106;
    float32 var_107;
    int32 var_108;
    float32 var_109;
    int32 var_110;
    float32 var_111;
    const int32 var_112 = 4;
    int32 var_113;
    float32 var_114;
    const int32 var_115 = 5;
    int32 var_116;
    float32 var_117;
    const int32 var_118 = 6;
    int32 var_119;
    float32 var_120;
    int32 var_121;
    float32 var_122;
    int32 var_123;
    float32 var_124;
    int32 var_125;
    float32 var_126;
    int32 var_127;
    float32 var_128;
    int32 var_129;
    float32 var_130;
    int32 var_131;
    quat var_132;
    const int32 var_133 = 5;
    bool var_134;
    quat var_135;
    quat var_136;
    quat var_137;
    quat var_138;
    quat var_139;
    quat var_140;
    vec3 var_141;
    const float32 var_142 = 1.0;
    const float32 var_143 = 0.0;
    vec3 var_144;
    float32 var_145;
    quat var_146;
    vec3 var_147;
    vec3 var_148;
    float32 var_149;
    quat var_150;
    quat var_151;
    vec3 var_152;
    vec3 var_153;
    float32 var_154;
    quat var_155;
    quat var_156;
    float32 var_157;
    int32 var_158;
    float32 var_159;
    int32 var_160;
    float32 var_161;
    int32 var_162;
    vec3 var_163;
    float32 var_164;
    int32 var_165;
    vec3 var_166;
    float32 var_167;
    int32 var_168;
    vec3 var_169;
    float32 var_170;
    int32 var_171;
    quat var_172;
    const int32 var_173 = 6;
    bool var_174;
    quat var_175;
    quat var_176;
    quat var_177;
    quat var_178;
    quat var_179;
    quat var_180;
    vec3 var_181;
    vec3 var_182;
    float32 var_183;
    quat var_184;
    vec3 var_185;
    vec3 var_186;
    float32 var_187;
    quat var_188;
    quat var_189;
    float32 var_190;
    int32 var_191;
    float32 var_192;
    int32 var_193;
    vec3 var_194;
    float32 var_195;
    int32 var_196;
    vec3 var_197;
    float32 var_198;
    int32 var_199;
    quat var_200;
    quat var_201;
    vec3 var_202;
    vec3 var_203;
    quat var_204;
    vec3 var_205;
    quat var_206;
    quat var_207;
    //---------
    // dual vars
    int32 adj_0 = 0;
    int32 adj_1 = 0;
    int32 adj_2 = 0;
    transform adj_3 = 0;
    transform adj_4 = 0;
    transform adj_5 = 0;
    vec3 adj_6 = 0;
    vec3 adj_7 = 0;
    vec3 adj_8 = 0;
    int32 adj_9 = 0;
    bool adj_10 = 0;
    transform adj_11 = 0;
    transform adj_12 = 0;
    vec3 adj_13 = 0;
    transform adj_14 = 0;
    vec3 adj_15 = 0;
    vec3 adj_16 = 0;
    vec3 adj_17 = 0;
    spatial_vector adj_18 = 0;
    vec3 adj_19 = 0;
    vec3 adj_20 = 0;
    vec3 adj_21 = 0;
    vec3 adj_22 = 0;
    transform adj_23 = 0;
    vec3 adj_24 = 0;
    vec3 adj_25 = 0;
    transform adj_26 = 0;
    vec3 adj_27 = 0;
    transform adj_28 = 0;
    vec3 adj_29 = 0;
    vec3 adj_30 = 0;
    vec3 adj_31 = 0;
    spatial_vector adj_32 = 0;
    vec3 adj_33 = 0;
    vec3 adj_34 = 0;
    vec3 adj_35 = 0;
    vec3 adj_36 = 0;
    int32 adj_37 = 0;
    vec3 adj_38 = 0;
    vec3 adj_39 = 0;
    vec3 adj_40 = 0;
    quat adj_41 = 0;
    quat adj_42 = 0;
    vec3 adj_43 = 0;
    vec3 adj_44 = 0;
    vec3 adj_45 = 0;
    int32 adj_46 = 0;
    int32 adj_47 = 0;
    int32 adj_48 = 0;
    bool adj_49 = 0;
    vec3 adj_50 = 0;
    float32 adj_51 = 0;
    float32 adj_52 = 0;
    int32 adj_53 = 0;
    bool adj_54 = 0;
    vec3 adj_55 = 0;
    vec3 adj_56 = 0;
    quat adj_57 = 0;
    quat adj_58 = 0;
    quat adj_59 = 0;
    int32 adj_60 = 0;
    float32 adj_61 = 0;
    float32 adj_62 = 0;
    float32 adj_63 = 0;
    float32 adj_64 = 0;
    float32 adj_65 = 0;
    int32 adj_66 = 0;
    float32 adj_67 = 0;
    int32 adj_68 = 0;
    float32 adj_69 = 0;
    vec3 adj_70 = 0;
    float32 adj_71 = 0;
    float32 adj_72 = 0;
    float32 adj_73 = 0;
    float32 adj_74 = 0;
    vec3 adj_75 = 0;
    float32 adj_76 = 0;
    float32 adj_77 = 0;
    int32 adj_78 = 0;
    bool adj_79 = 0;
    quat adj_80 = 0;
    quat adj_81 = 0;
    float32 adj_82 = 0;
    int32 adj_83 = 0;
    float32 adj_84 = 0;
    int32 adj_85 = 0;
    float32 adj_86 = 0;
    int32 adj_87 = 0;
    float32 adj_88 = 0;
    int32 adj_89 = 0;
    float32 adj_90 = 0;
    int32 adj_91 = 0;
    float32 adj_92 = 0;
    int32 adj_93 = 0;
    float32 adj_94 = 0;
    int32 adj_95 = 0;
    quat adj_96 = 0;
    int32 adj_97 = 0;
    bool adj_98 = 0;
    int32 adj_99 = 0;
    bool adj_100 = 0;
    quat adj_101 = 0;
    quat adj_102 = 0;
    float32 adj_103 = 0;
    int32 adj_104 = 0;
    float32 adj_105 = 0;
    int32 adj_106 = 0;
    float32 adj_107 = 0;
    int32 adj_108 = 0;
    float32 adj_109 = 0;
    int32 adj_110 = 0;
    float32 adj_111 = 0;
    int32 adj_112 = 0;
    int32 adj_113 = 0;
    float32 adj_114 = 0;
    int32 adj_115 = 0;
    int32 adj_116 = 0;
    float32 adj_117 = 0;
    int32 adj_118 = 0;
    int32 adj_119 = 0;
    float32 adj_120 = 0;
    int32 adj_121 = 0;
    float32 adj_122 = 0;
    int32 adj_123 = 0;
    float32 adj_124 = 0;
    int32 adj_125 = 0;
    float32 adj_126 = 0;
    int32 adj_127 = 0;
    float32 adj_128 = 0;
    int32 adj_129 = 0;
    float32 adj_130 = 0;
    int32 adj_131 = 0;
    quat adj_132 = 0;
    int32 adj_133 = 0;
    bool adj_134 = 0;
    quat adj_135 = 0;
    quat adj_136 = 0;
    quat adj_137 = 0;
    quat adj_138 = 0;
    quat adj_139 = 0;
    quat adj_140 = 0;
    vec3 adj_141 = 0;
    float32 adj_142 = 0;
    float32 adj_143 = 0;
    vec3 adj_144 = 0;
    float32 adj_145 = 0;
    quat adj_146 = 0;
    vec3 adj_147 = 0;
    vec3 adj_148 = 0;
    float32 adj_149 = 0;
    quat adj_150 = 0;
    quat adj_151 = 0;
    vec3 adj_152 = 0;
    vec3 adj_153 = 0;
    float32 adj_154 = 0;
    quat adj_155 = 0;
    quat adj_156 = 0;
    float32 adj_157 = 0;
    int32 adj_158 = 0;
    float32 adj_159 = 0;
    int32 adj_160 = 0;
    float32 adj_161 = 0;
    int32 adj_162 = 0;
    vec3 adj_163 = 0;
    float32 adj_164 = 0;
    int32 adj_165 = 0;
    vec3 adj_166 = 0;
    float32 adj_167 = 0;
    int32 adj_168 = 0;
    vec3 adj_169 = 0;
    float32 adj_170 = 0;
    int32 adj_171 = 0;
    quat adj_172 = 0;
    int32 adj_173 = 0;
    bool adj_174 = 0;
    quat adj_175 = 0;
    quat adj_176 = 0;
    quat adj_177 = 0;
    quat adj_178 = 0;
    quat adj_179 = 0;
    quat adj_180 = 0;
    vec3 adj_181 = 0;
    vec3 adj_182 = 0;
    float32 adj_183 = 0;
    quat adj_184 = 0;
    vec3 adj_185 = 0;
    vec3 adj_186 = 0;
    float32 adj_187 = 0;
    quat adj_188 = 0;
    quat adj_189 = 0;
    float32 adj_190 = 0;
    int32 adj_191 = 0;
    float32 adj_192 = 0;
    int32 adj_193 = 0;
    vec3 adj_194 = 0;
    float32 adj_195 = 0;
    int32 adj_196 = 0;
    vec3 adj_197 = 0;
    float32 adj_198 = 0;
    int32 adj_199 = 0;
    quat adj_200 = 0;
    quat adj_201 = 0;
    vec3 adj_202 = 0;
    vec3 adj_203 = 0;
    quat adj_204 = 0;
    vec3 adj_205 = 0;
    quat adj_206 = 0;
    quat adj_207 = 0;
        //---------
        // forward
        var_0 = wp::tid();
        wp::copy(var_1, var_0);
        var_2 = wp::load(var_joint_parent, var_0);
        var_3 = wp::load(var_joint_X_p, var_0);
        var_4 = wp::load(var_joint_X_c, var_0);
        wp::copy(var_5, var_3);
        var_6 = wp::vec3();
        var_7 = wp::vec3();
        var_8 = wp::vec3();
        var_10 = (var_2 >= var_9);
        if (var_10) {
        	var_11 = wp::load(var_body_q, var_2);
        	var_12 = wp::mul(var_11, var_5);
        	var_13 = wp::transform_get_translation(var_12);
        	var_14 = wp::load(var_body_q, var_2);
        	var_15 = wp::load(var_body_com, var_2);
        	var_16 = wp::transform_point(var_14, var_15);
        	var_17 = wp::sub(var_13, var_16);
        	var_18 = wp::load(var_body_qd, var_2);
        	var_19 = wp::spatial_top(var_18);
        	var_20 = wp::spatial_bottom(var_18);
        	var_21 = wp::cross(var_19, var_17);
        	var_22 = wp::add(var_20, var_21);
        }
        var_23 = wp::select(var_10, var_5, var_12);
        var_24 = wp::select(var_10, var_7, var_19);
        var_25 = wp::select(var_10, var_8, var_22);
        var_26 = wp::load(var_body_q, var_1);
        var_27 = wp::transform_get_translation(var_26);
        var_28 = wp::load(var_body_q, var_1);
        var_29 = wp::load(var_body_com, var_1);
        var_30 = wp::transform_point(var_28, var_29);
        var_31 = wp::sub(var_27, var_30);
        var_32 = wp::load(var_body_qd, var_1);
        var_33 = wp::spatial_top(var_32);
        var_34 = wp::spatial_bottom(var_32);
        var_35 = wp::cross(var_33, var_31);
        var_36 = wp::add(var_34, var_35);
        var_37 = wp::load(var_joint_type, var_0);
        var_38 = wp::load(var_joint_axis, var_0);
        var_39 = wp::transform_get_translation(var_23);
        var_40 = wp::transform_get_translation(var_26);
        var_41 = wp::transform_get_rotation(var_23);
        var_42 = wp::transform_get_rotation(var_26);
        var_43 = wp::sub(var_40, var_39);
        var_44 = wp::sub(var_36, var_25);
        var_45 = wp::sub(var_33, var_24);
        var_46 = wp::load(var_joint_q_start, var_0);
        var_47 = wp::load(var_joint_qd_start, var_0);
        var_49 = (var_37 == var_48);
        if (var_49) {
        	var_50 = wp::transform_vector(var_23, var_38);
        	var_51 = wp::dot(var_43, var_50);
        	var_52 = wp::dot(var_44, var_50);
        	//wp::store(var_joint_q, var_46, var_51);
        	//wp::store(var_joint_qd, var_47, var_52);
        	goto label0;
        }
        var_54 = (var_37 == var_53);
        if (var_54) {
        	var_55 = wp::transform_vector(var_23, var_38);
        	var_56 = wp::transform_vector(var_26, var_38);
        	var_57 = wp::quat_inverse(var_41);
        	var_58 = wp::mul(var_57, var_42);
        	var_59 = quat_twist(var_38, var_58);
        	var_61 = wp::index(var_59, var_60);
        	var_62 = wp::acos(var_61);
        	var_64 = wp::mul(var_62, var_63);
        	var_65 = wp::index(var_59, var_9);
        	var_67 = wp::index(var_59, var_66);
        	var_69 = wp::index(var_59, var_68);
        	var_70 = wp::vec3(var_65, var_67, var_69);
        	var_71 = wp::dot(var_38, var_70);
        	var_72 = wp::sign(var_71);
        	var_73 = wp::mul(var_64, var_72);
        	var_74 = wp::dot(var_45, var_55);
        	//wp::store(var_joint_q, var_46, var_73);
        	//wp::store(var_joint_qd, var_47, var_74);
        	goto label1;
        }
        var_75 = wp::select(var_54, var_50, var_55);
        var_76 = wp::select(var_54, var_51, var_73);
        var_77 = wp::select(var_54, var_52, var_74);
        var_79 = (var_37 == var_78);
        if (var_79) {
        	var_80 = wp::quat_inverse(var_41);
        	var_81 = wp::mul(var_80, var_42);
        	var_82 = wp::index(var_81, var_9);
        	var_83 = wp::add(var_46, var_9);
        	//wp::store(var_joint_q, var_83, var_82);
        	var_84 = wp::index(var_81, var_66);
        	var_85 = wp::add(var_46, var_66);
        	//wp::store(var_joint_q, var_85, var_84);
        	var_86 = wp::index(var_81, var_68);
        	var_87 = wp::add(var_46, var_68);
        	//wp::store(var_joint_q, var_87, var_86);
        	var_88 = wp::index(var_81, var_60);
        	var_89 = wp::add(var_46, var_60);
        	//wp::store(var_joint_q, var_89, var_88);
        	var_90 = wp::index(var_45, var_9);
        	var_91 = wp::add(var_47, var_9);
        	//wp::store(var_joint_qd, var_91, var_90);
        	var_92 = wp::index(var_45, var_66);
        	var_93 = wp::add(var_47, var_66);
        	//wp::store(var_joint_qd, var_93, var_92);
        	var_94 = wp::index(var_45, var_68);
        	var_95 = wp::add(var_47, var_68);
        	//wp::store(var_joint_qd, var_95, var_94);
        	goto label2;
        }
        var_96 = wp::select(var_79, var_58, var_81);
        var_98 = (var_37 == var_97);
        if (var_98) {
        	goto label3;
        }
        var_100 = (var_37 == var_99);
        if (var_100) {
        	var_101 = wp::quat_inverse(var_41);
        	var_102 = wp::mul(var_101, var_42);
        	var_103 = wp::index(var_43, var_9);
        	var_104 = wp::add(var_46, var_9);
        	//wp::store(var_joint_q, var_104, var_103);
        	var_105 = wp::index(var_43, var_66);
        	var_106 = wp::add(var_46, var_66);
        	//wp::store(var_joint_q, var_106, var_105);
        	var_107 = wp::index(var_43, var_68);
        	var_108 = wp::add(var_46, var_68);
        	//wp::store(var_joint_q, var_108, var_107);
        	var_109 = wp::index(var_102, var_9);
        	var_110 = wp::add(var_46, var_60);
        	//wp::store(var_joint_q, var_110, var_109);
        	var_111 = wp::index(var_102, var_66);
        	var_113 = wp::add(var_46, var_112);
        	//wp::store(var_joint_q, var_113, var_111);
        	var_114 = wp::index(var_102, var_68);
        	var_116 = wp::add(var_46, var_115);
        	//wp::store(var_joint_q, var_116, var_114);
        	var_117 = wp::index(var_102, var_60);
        	var_119 = wp::add(var_46, var_118);
        	//wp::store(var_joint_q, var_119, var_117);
        	var_120 = wp::index(var_45, var_9);
        	var_121 = wp::add(var_47, var_9);
        	//wp::store(var_joint_qd, var_121, var_120);
        	var_122 = wp::index(var_45, var_66);
        	var_123 = wp::add(var_47, var_66);
        	//wp::store(var_joint_qd, var_123, var_122);
        	var_124 = wp::index(var_45, var_68);
        	var_125 = wp::add(var_47, var_68);
        	//wp::store(var_joint_qd, var_125, var_124);
        	var_126 = wp::index(var_44, var_9);
        	var_127 = wp::add(var_47, var_60);
        	//wp::store(var_joint_qd, var_127, var_126);
        	var_128 = wp::index(var_44, var_66);
        	var_129 = wp::add(var_47, var_112);
        	//wp::store(var_joint_qd, var_129, var_128);
        	var_130 = wp::index(var_44, var_68);
        	var_131 = wp::add(var_47, var_115);
        	//wp::store(var_joint_qd, var_131, var_130);
        }
        var_132 = wp::select(var_100, var_96, var_102);
        var_134 = (var_37 == var_133);
        if (var_134) {
        	var_135 = wp::transform_get_rotation(var_4);
        	var_136 = wp::quat_inverse(var_135);
        	var_137 = wp::quat_inverse(var_41);
        	var_138 = wp::mul(var_136, var_137);
        	var_139 = wp::mul(var_138, var_42);
        	var_140 = wp::mul(var_139, var_135);
        	var_141 = quat_decompose(var_140);
        	var_144 = wp::vec3(var_142, var_143, var_143);
        	var_145 = wp::index(var_141, var_9);
        	var_146 = wp::quat_from_axis_angle(var_144, var_145);
        	var_147 = wp::vec3(var_143, var_142, var_143);
        	var_148 = wp::quat_rotate(var_146, var_147);
        	var_149 = wp::index(var_141, var_66);
        	var_150 = wp::quat_from_axis_angle(var_148, var_149);
        	var_151 = wp::mul(var_150, var_146);
        	var_152 = wp::vec3(var_143, var_143, var_142);
        	var_153 = wp::quat_rotate(var_151, var_152);
        	var_154 = wp::index(var_141, var_68);
        	var_155 = wp::quat_from_axis_angle(var_153, var_154);
        	var_156 = wp::mul(var_41, var_135);
        	var_157 = wp::index(var_141, var_9);
        	var_158 = wp::add(var_46, var_9);
        	//wp::store(var_joint_q, var_158, var_157);
        	var_159 = wp::index(var_141, var_66);
        	var_160 = wp::add(var_46, var_66);
        	//wp::store(var_joint_q, var_160, var_159);
        	var_161 = wp::index(var_141, var_68);
        	var_162 = wp::add(var_46, var_68);
        	//wp::store(var_joint_q, var_162, var_161);
        	var_163 = wp::quat_rotate(var_156, var_144);
        	var_164 = wp::dot(var_163, var_45);
        	var_165 = wp::add(var_47, var_9);
        	//wp::store(var_joint_qd, var_165, var_164);
        	var_166 = wp::quat_rotate(var_156, var_148);
        	var_167 = wp::dot(var_166, var_45);
        	var_168 = wp::add(var_47, var_66);
        	//wp::store(var_joint_qd, var_168, var_167);
        	var_169 = wp::quat_rotate(var_156, var_153);
        	var_170 = wp::dot(var_169, var_45);
        	var_171 = wp::add(var_47, var_68);
        	//wp::store(var_joint_qd, var_171, var_170);
        	goto label4;
        }
        var_172 = wp::select(var_134, var_132, var_140);
        var_174 = (var_37 == var_173);
        if (var_174) {
        	var_175 = wp::transform_get_rotation(var_4);
        	var_176 = wp::quat_inverse(var_175);
        	var_177 = wp::quat_inverse(var_41);
        	var_178 = wp::mul(var_176, var_177);
        	var_179 = wp::mul(var_178, var_42);
        	var_180 = wp::mul(var_179, var_175);
        	var_181 = quat_decompose(var_180);
        	var_182 = wp::vec3(var_142, var_143, var_143);
        	var_183 = wp::index(var_181, var_9);
        	var_184 = wp::quat_from_axis_angle(var_182, var_183);
        	var_185 = wp::vec3(var_143, var_142, var_143);
        	var_186 = wp::quat_rotate(var_184, var_185);
        	var_187 = wp::index(var_181, var_66);
        	var_188 = wp::quat_from_axis_angle(var_186, var_187);
        	var_189 = wp::mul(var_41, var_175);
        	var_190 = wp::index(var_181, var_9);
        	var_191 = wp::add(var_46, var_9);
        	//wp::store(var_joint_q, var_191, var_190);
        	var_192 = wp::index(var_181, var_66);
        	var_193 = wp::add(var_46, var_66);
        	//wp::store(var_joint_q, var_193, var_192);
        	var_194 = wp::quat_rotate(var_189, var_182);
        	var_195 = wp::dot(var_194, var_45);
        	var_196 = wp::add(var_47, var_9);
        	//wp::store(var_joint_qd, var_196, var_195);
        	var_197 = wp::quat_rotate(var_189, var_186);
        	var_198 = wp::dot(var_197, var_45);
        	var_199 = wp::add(var_47, var_66);
        	//wp::store(var_joint_qd, var_199, var_198);
        	goto label5;
        }
        var_200 = wp::select(var_174, var_172, var_180);
        var_201 = wp::select(var_174, var_135, var_175);
        var_202 = wp::select(var_174, var_141, var_181);
        var_203 = wp::select(var_174, var_144, var_182);
        var_204 = wp::select(var_174, var_146, var_184);
        var_205 = wp::select(var_174, var_148, var_186);
        var_206 = wp::select(var_174, var_150, var_188);
        var_207 = wp::select(var_174, var_156, var_189);
        //---------
        // reverse
        wp::adj_select(var_174, var_156, var_189, adj_174, adj_156, adj_189, adj_207);
        wp::adj_select(var_174, var_150, var_188, adj_174, adj_150, adj_188, adj_206);
        wp::adj_select(var_174, var_148, var_186, adj_174, adj_148, adj_186, adj_205);
        wp::adj_select(var_174, var_146, var_184, adj_174, adj_146, adj_184, adj_204);
        wp::adj_select(var_174, var_144, var_182, adj_174, adj_144, adj_182, adj_203);
        wp::adj_select(var_174, var_141, var_181, adj_174, adj_141, adj_181, adj_202);
        wp::adj_select(var_174, var_135, var_175, adj_174, adj_135, adj_175, adj_201);
        wp::adj_select(var_174, var_172, var_180, adj_174, adj_172, adj_180, adj_200);
        if (var_174) {
        	label5:;
        	wp::adj_store(var_joint_qd, var_199, var_198, adj_joint_qd, adj_199, adj_198);
        	wp::adj_add(var_47, var_66, adj_47, adj_66, adj_199);
        	wp::adj_dot(var_197, var_45, adj_197, adj_45, adj_198);
        	wp::adj_quat_rotate(var_189, var_186, adj_189, adj_186, adj_197);
        	wp::adj_store(var_joint_qd, var_196, var_195, adj_joint_qd, adj_196, adj_195);
        	wp::adj_add(var_47, var_9, adj_47, adj_9, adj_196);
        	wp::adj_dot(var_194, var_45, adj_194, adj_45, adj_195);
        	wp::adj_quat_rotate(var_189, var_182, adj_189, adj_182, adj_194);
        	wp::adj_store(var_joint_q, var_193, var_192, adj_joint_q, adj_193, adj_192);
        	wp::adj_add(var_46, var_66, adj_46, adj_66, adj_193);
        	wp::adj_index(var_181, var_66, adj_181, adj_66, adj_192);
        	wp::adj_store(var_joint_q, var_191, var_190, adj_joint_q, adj_191, adj_190);
        	wp::adj_add(var_46, var_9, adj_46, adj_9, adj_191);
        	wp::adj_index(var_181, var_9, adj_181, adj_9, adj_190);
        	wp::adj_mul(var_41, var_175, adj_41, adj_175, adj_189);
        	wp::adj_quat_from_axis_angle(var_186, var_187, adj_186, adj_187, adj_188);
        	wp::adj_index(var_181, var_66, adj_181, adj_66, adj_187);
        	wp::adj_quat_rotate(var_184, var_185, adj_184, adj_185, adj_186);
        	wp::adj_vec3(var_143, var_142, var_143, adj_143, adj_142, adj_143, adj_185);
        	wp::adj_quat_from_axis_angle(var_182, var_183, adj_182, adj_183, adj_184);
        	wp::adj_index(var_181, var_9, adj_181, adj_9, adj_183);
        	wp::adj_vec3(var_142, var_143, var_143, adj_142, adj_143, adj_143, adj_182);
        	adj_quat_decompose(var_180, adj_180, adj_181);
        	wp::adj_mul(var_179, var_175, adj_179, adj_175, adj_180);
        	wp::adj_mul(var_178, var_42, adj_178, adj_42, adj_179);
        	wp::adj_mul(var_176, var_177, adj_176, adj_177, adj_178);
        	wp::adj_quat_inverse(var_41, adj_41, adj_177);
        	wp::adj_quat_inverse(var_175, adj_175, adj_176);
        	wp::adj_transform_get_rotation(var_4, adj_4, adj_175);
        }
        wp::adj_select(var_134, var_132, var_140, adj_134, adj_132, adj_140, adj_172);
        if (var_134) {
        	label4:;
        	wp::adj_store(var_joint_qd, var_171, var_170, adj_joint_qd, adj_171, adj_170);
        	wp::adj_add(var_47, var_68, adj_47, adj_68, adj_171);
        	wp::adj_dot(var_169, var_45, adj_169, adj_45, adj_170);
        	wp::adj_quat_rotate(var_156, var_153, adj_156, adj_153, adj_169);
        	wp::adj_store(var_joint_qd, var_168, var_167, adj_joint_qd, adj_168, adj_167);
        	wp::adj_add(var_47, var_66, adj_47, adj_66, adj_168);
        	wp::adj_dot(var_166, var_45, adj_166, adj_45, adj_167);
        	wp::adj_quat_rotate(var_156, var_148, adj_156, adj_148, adj_166);
        	wp::adj_store(var_joint_qd, var_165, var_164, adj_joint_qd, adj_165, adj_164);
        	wp::adj_add(var_47, var_9, adj_47, adj_9, adj_165);
        	wp::adj_dot(var_163, var_45, adj_163, adj_45, adj_164);
        	wp::adj_quat_rotate(var_156, var_144, adj_156, adj_144, adj_163);
        	wp::adj_store(var_joint_q, var_162, var_161, adj_joint_q, adj_162, adj_161);
        	wp::adj_add(var_46, var_68, adj_46, adj_68, adj_162);
        	wp::adj_index(var_141, var_68, adj_141, adj_68, adj_161);
        	wp::adj_store(var_joint_q, var_160, var_159, adj_joint_q, adj_160, adj_159);
        	wp::adj_add(var_46, var_66, adj_46, adj_66, adj_160);
        	wp::adj_index(var_141, var_66, adj_141, adj_66, adj_159);
        	wp::adj_store(var_joint_q, var_158, var_157, adj_joint_q, adj_158, adj_157);
        	wp::adj_add(var_46, var_9, adj_46, adj_9, adj_158);
        	wp::adj_index(var_141, var_9, adj_141, adj_9, adj_157);
        	wp::adj_mul(var_41, var_135, adj_41, adj_135, adj_156);
        	wp::adj_quat_from_axis_angle(var_153, var_154, adj_153, adj_154, adj_155);
        	wp::adj_index(var_141, var_68, adj_141, adj_68, adj_154);
        	wp::adj_quat_rotate(var_151, var_152, adj_151, adj_152, adj_153);
        	wp::adj_vec3(var_143, var_143, var_142, adj_143, adj_143, adj_142, adj_152);
        	wp::adj_mul(var_150, var_146, adj_150, adj_146, adj_151);
        	wp::adj_quat_from_axis_angle(var_148, var_149, adj_148, adj_149, adj_150);
        	wp::adj_index(var_141, var_66, adj_141, adj_66, adj_149);
        	wp::adj_quat_rotate(var_146, var_147, adj_146, adj_147, adj_148);
        	wp::adj_vec3(var_143, var_142, var_143, adj_143, adj_142, adj_143, adj_147);
        	wp::adj_quat_from_axis_angle(var_144, var_145, adj_144, adj_145, adj_146);
        	wp::adj_index(var_141, var_9, adj_141, adj_9, adj_145);
        	wp::adj_vec3(var_142, var_143, var_143, adj_142, adj_143, adj_143, adj_144);
        	adj_quat_decompose(var_140, adj_140, adj_141);
        	wp::adj_mul(var_139, var_135, adj_139, adj_135, adj_140);
        	wp::adj_mul(var_138, var_42, adj_138, adj_42, adj_139);
        	wp::adj_mul(var_136, var_137, adj_136, adj_137, adj_138);
        	wp::adj_quat_inverse(var_41, adj_41, adj_137);
        	wp::adj_quat_inverse(var_135, adj_135, adj_136);
        	wp::adj_transform_get_rotation(var_4, adj_4, adj_135);
        }
        wp::adj_select(var_100, var_96, var_102, adj_100, adj_96, adj_102, adj_132);
        if (var_100) {
        	wp::adj_store(var_joint_qd, var_131, var_130, adj_joint_qd, adj_131, adj_130);
        	wp::adj_add(var_47, var_115, adj_47, adj_115, adj_131);
        	wp::adj_index(var_44, var_68, adj_44, adj_68, adj_130);
        	wp::adj_store(var_joint_qd, var_129, var_128, adj_joint_qd, adj_129, adj_128);
        	wp::adj_add(var_47, var_112, adj_47, adj_112, adj_129);
        	wp::adj_index(var_44, var_66, adj_44, adj_66, adj_128);
        	wp::adj_store(var_joint_qd, var_127, var_126, adj_joint_qd, adj_127, adj_126);
        	wp::adj_add(var_47, var_60, adj_47, adj_60, adj_127);
        	wp::adj_index(var_44, var_9, adj_44, adj_9, adj_126);
        	wp::adj_store(var_joint_qd, var_125, var_124, adj_joint_qd, adj_125, adj_124);
        	wp::adj_add(var_47, var_68, adj_47, adj_68, adj_125);
        	wp::adj_index(var_45, var_68, adj_45, adj_68, adj_124);
        	wp::adj_store(var_joint_qd, var_123, var_122, adj_joint_qd, adj_123, adj_122);
        	wp::adj_add(var_47, var_66, adj_47, adj_66, adj_123);
        	wp::adj_index(var_45, var_66, adj_45, adj_66, adj_122);
        	wp::adj_store(var_joint_qd, var_121, var_120, adj_joint_qd, adj_121, adj_120);
        	wp::adj_add(var_47, var_9, adj_47, adj_9, adj_121);
        	wp::adj_index(var_45, var_9, adj_45, adj_9, adj_120);
        	wp::adj_store(var_joint_q, var_119, var_117, adj_joint_q, adj_119, adj_117);
        	wp::adj_add(var_46, var_118, adj_46, adj_118, adj_119);
        	wp::adj_index(var_102, var_60, adj_102, adj_60, adj_117);
        	wp::adj_store(var_joint_q, var_116, var_114, adj_joint_q, adj_116, adj_114);
        	wp::adj_add(var_46, var_115, adj_46, adj_115, adj_116);
        	wp::adj_index(var_102, var_68, adj_102, adj_68, adj_114);
        	wp::adj_store(var_joint_q, var_113, var_111, adj_joint_q, adj_113, adj_111);
        	wp::adj_add(var_46, var_112, adj_46, adj_112, adj_113);
        	wp::adj_index(var_102, var_66, adj_102, adj_66, adj_111);
        	wp::adj_store(var_joint_q, var_110, var_109, adj_joint_q, adj_110, adj_109);
        	wp::adj_add(var_46, var_60, adj_46, adj_60, adj_110);
        	wp::adj_index(var_102, var_9, adj_102, adj_9, adj_109);
        	wp::adj_store(var_joint_q, var_108, var_107, adj_joint_q, adj_108, adj_107);
        	wp::adj_add(var_46, var_68, adj_46, adj_68, adj_108);
        	wp::adj_index(var_43, var_68, adj_43, adj_68, adj_107);
        	wp::adj_store(var_joint_q, var_106, var_105, adj_joint_q, adj_106, adj_105);
        	wp::adj_add(var_46, var_66, adj_46, adj_66, adj_106);
        	wp::adj_index(var_43, var_66, adj_43, adj_66, adj_105);
        	wp::adj_store(var_joint_q, var_104, var_103, adj_joint_q, adj_104, adj_103);
        	wp::adj_add(var_46, var_9, adj_46, adj_9, adj_104);
        	wp::adj_index(var_43, var_9, adj_43, adj_9, adj_103);
        	wp::adj_mul(var_101, var_42, adj_101, adj_42, adj_102);
        	wp::adj_quat_inverse(var_41, adj_41, adj_101);
        }
        if (var_98) {
        	label3:;
        }
        wp::adj_select(var_79, var_58, var_81, adj_79, adj_58, adj_81, adj_96);
        if (var_79) {
        	label2:;
        	wp::adj_store(var_joint_qd, var_95, var_94, adj_joint_qd, adj_95, adj_94);
        	wp::adj_add(var_47, var_68, adj_47, adj_68, adj_95);
        	wp::adj_index(var_45, var_68, adj_45, adj_68, adj_94);
        	wp::adj_store(var_joint_qd, var_93, var_92, adj_joint_qd, adj_93, adj_92);
        	wp::adj_add(var_47, var_66, adj_47, adj_66, adj_93);
        	wp::adj_index(var_45, var_66, adj_45, adj_66, adj_92);
        	wp::adj_store(var_joint_qd, var_91, var_90, adj_joint_qd, adj_91, adj_90);
        	wp::adj_add(var_47, var_9, adj_47, adj_9, adj_91);
        	wp::adj_index(var_45, var_9, adj_45, adj_9, adj_90);
        	wp::adj_store(var_joint_q, var_89, var_88, adj_joint_q, adj_89, adj_88);
        	wp::adj_add(var_46, var_60, adj_46, adj_60, adj_89);
        	wp::adj_index(var_81, var_60, adj_81, adj_60, adj_88);
        	wp::adj_store(var_joint_q, var_87, var_86, adj_joint_q, adj_87, adj_86);
        	wp::adj_add(var_46, var_68, adj_46, adj_68, adj_87);
        	wp::adj_index(var_81, var_68, adj_81, adj_68, adj_86);
        	wp::adj_store(var_joint_q, var_85, var_84, adj_joint_q, adj_85, adj_84);
        	wp::adj_add(var_46, var_66, adj_46, adj_66, adj_85);
        	wp::adj_index(var_81, var_66, adj_81, adj_66, adj_84);
        	wp::adj_store(var_joint_q, var_83, var_82, adj_joint_q, adj_83, adj_82);
        	wp::adj_add(var_46, var_9, adj_46, adj_9, adj_83);
        	wp::adj_index(var_81, var_9, adj_81, adj_9, adj_82);
        	wp::adj_mul(var_80, var_42, adj_80, adj_42, adj_81);
        	wp::adj_quat_inverse(var_41, adj_41, adj_80);
        }
        wp::adj_select(var_54, var_52, var_74, adj_54, adj_52, adj_74, adj_77);
        wp::adj_select(var_54, var_51, var_73, adj_54, adj_51, adj_73, adj_76);
        wp::adj_select(var_54, var_50, var_55, adj_54, adj_50, adj_55, adj_75);
        if (var_54) {
        	label1:;
        	wp::adj_store(var_joint_qd, var_47, var_74, adj_joint_qd, adj_47, adj_74);
        	wp::adj_store(var_joint_q, var_46, var_73, adj_joint_q, adj_46, adj_73);
        	wp::adj_dot(var_45, var_55, adj_45, adj_55, adj_74);
        	wp::adj_mul(var_64, var_72, adj_64, adj_72, adj_73);
        	wp::adj_sign(var_71, adj_71, adj_72);
        	wp::adj_dot(var_38, var_70, adj_38, adj_70, adj_71);
        	wp::adj_vec3(var_65, var_67, var_69, adj_65, adj_67, adj_69, adj_70);
        	wp::adj_index(var_59, var_68, adj_59, adj_68, adj_69);
        	wp::adj_index(var_59, var_66, adj_59, adj_66, adj_67);
        	wp::adj_index(var_59, var_9, adj_59, adj_9, adj_65);
        	wp::adj_mul(var_62, var_63, adj_62, adj_63, adj_64);
        	wp::adj_acos(var_61, adj_61, adj_62);
        	wp::adj_index(var_59, var_60, adj_59, adj_60, adj_61);
        	adj_quat_twist(var_38, var_58, adj_38, adj_58, adj_59);
        	wp::adj_mul(var_57, var_42, adj_57, adj_42, adj_58);
        	wp::adj_quat_inverse(var_41, adj_41, adj_57);
        	wp::adj_transform_vector(var_26, var_38, adj_26, adj_38, adj_56);
        	wp::adj_transform_vector(var_23, var_38, adj_23, adj_38, adj_55);
        }
        if (var_49) {
        	label0:;
        	wp::adj_store(var_joint_qd, var_47, var_52, adj_joint_qd, adj_47, adj_52);
        	wp::adj_store(var_joint_q, var_46, var_51, adj_joint_q, adj_46, adj_51);
        	wp::adj_dot(var_44, var_50, adj_44, adj_50, adj_52);
        	wp::adj_dot(var_43, var_50, adj_43, adj_50, adj_51);
        	wp::adj_transform_vector(var_23, var_38, adj_23, adj_38, adj_50);
        }
        wp::adj_load(var_joint_qd_start, var_0, adj_joint_qd_start, adj_0, adj_47);
        wp::adj_load(var_joint_q_start, var_0, adj_joint_q_start, adj_0, adj_46);
        wp::adj_sub(var_33, var_24, adj_33, adj_24, adj_45);
        wp::adj_sub(var_36, var_25, adj_36, adj_25, adj_44);
        wp::adj_sub(var_40, var_39, adj_40, adj_39, adj_43);
        wp::adj_transform_get_rotation(var_26, adj_26, adj_42);
        wp::adj_transform_get_rotation(var_23, adj_23, adj_41);
        wp::adj_transform_get_translation(var_26, adj_26, adj_40);
        wp::adj_transform_get_translation(var_23, adj_23, adj_39);
        wp::adj_load(var_joint_axis, var_0, adj_joint_axis, adj_0, adj_38);
        wp::adj_load(var_joint_type, var_0, adj_joint_type, adj_0, adj_37);
        wp::adj_add(var_34, var_35, adj_34, adj_35, adj_36);
        wp::adj_cross(var_33, var_31, adj_33, adj_31, adj_35);
        wp::adj_spatial_bottom(var_32, adj_32, adj_34);
        wp::adj_spatial_top(var_32, adj_32, adj_33);
        wp::adj_load(var_body_qd, var_1, adj_body_qd, adj_1, adj_32);
        wp::adj_sub(var_27, var_30, adj_27, adj_30, adj_31);
        wp::adj_transform_point(var_28, var_29, adj_28, adj_29, adj_30);
        wp::adj_load(var_body_com, var_1, adj_body_com, adj_1, adj_29);
        wp::adj_load(var_body_q, var_1, adj_body_q, adj_1, adj_28);
        wp::adj_transform_get_translation(var_26, adj_26, adj_27);
        wp::adj_load(var_body_q, var_1, adj_body_q, adj_1, adj_26);
        wp::adj_select(var_10, var_8, var_22, adj_10, adj_8, adj_22, adj_25);
        wp::adj_select(var_10, var_7, var_19, adj_10, adj_7, adj_19, adj_24);
        wp::adj_select(var_10, var_5, var_12, adj_10, adj_5, adj_12, adj_23);
        if (var_10) {
        	wp::adj_add(var_20, var_21, adj_20, adj_21, adj_22);
        	wp::adj_cross(var_19, var_17, adj_19, adj_17, adj_21);
        	wp::adj_spatial_bottom(var_18, adj_18, adj_20);
        	wp::adj_spatial_top(var_18, adj_18, adj_19);
        	wp::adj_load(var_body_qd, var_2, adj_body_qd, adj_2, adj_18);
        	wp::adj_sub(var_13, var_16, adj_13, adj_16, adj_17);
        	wp::adj_transform_point(var_14, var_15, adj_14, adj_15, adj_16);
        	wp::adj_load(var_body_com, var_2, adj_body_com, adj_2, adj_15);
        	wp::adj_load(var_body_q, var_2, adj_body_q, adj_2, adj_14);
        	wp::adj_transform_get_translation(var_12, adj_12, adj_13);
        	wp::adj_mul(var_11, var_5, adj_11, adj_5, adj_12);
        	wp::adj_load(var_body_q, var_2, adj_body_q, adj_2, adj_11);
        }
        wp::adj_copy(var_5, var_3, adj_5, adj_3);
        wp::adj_load(var_joint_X_c, var_0, adj_joint_X_c, adj_0, adj_4);
        wp::adj_load(var_joint_X_p, var_0, adj_joint_X_p, adj_0, adj_3);
        wp::adj_load(var_joint_parent, var_0, adj_joint_parent, adj_0, adj_2);
        wp::adj_copy(var_1, var_0, adj_1, adj_0);
        return;

}



extern "C" {

// Python entry points
WP_API void eval_articulation_ik_cuda_forward(void* stream, launch_bounds_t dim,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd,
	array_t<vec3> var_body_com,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd)
{
    eval_articulation_ik_cuda_kernel_forward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_body_q,
			var_body_qd,
			var_body_com,
			var_joint_type,
			var_joint_parent,
			var_joint_X_p,
			var_joint_X_c,
			var_joint_axis,
			var_joint_q_start,
			var_joint_qd_start,
			var_joint_q,
			var_joint_qd);
}

WP_API void eval_articulation_ik_cuda_backward(void* stream, launch_bounds_t dim,
	array_t<transform> var_body_q,
	array_t<spatial_vector> var_body_qd,
	array_t<vec3> var_body_com,
	array_t<int32> var_joint_type,
	array_t<int32> var_joint_parent,
	array_t<transform> var_joint_X_p,
	array_t<transform> var_joint_X_c,
	array_t<vec3> var_joint_axis,
	array_t<int32> var_joint_q_start,
	array_t<int32> var_joint_qd_start,
	array_t<float32> var_joint_q,
	array_t<float32> var_joint_qd,
	array_t<transform> adj_body_q,
	array_t<spatial_vector> adj_body_qd,
	array_t<vec3> adj_body_com,
	array_t<int32> adj_joint_type,
	array_t<int32> adj_joint_parent,
	array_t<transform> adj_joint_X_p,
	array_t<transform> adj_joint_X_c,
	array_t<vec3> adj_joint_axis,
	array_t<int32> adj_joint_q_start,
	array_t<int32> adj_joint_qd_start,
	array_t<float32> adj_joint_q,
	array_t<float32> adj_joint_qd)
{
    eval_articulation_ik_cuda_kernel_backward<<<(dim.size + 256 - 1) / 256, 256, 0, (hipStream_t)stream>>>(dim,
			var_body_q,
			var_body_qd,
			var_body_com,
			var_joint_type,
			var_joint_parent,
			var_joint_X_p,
			var_joint_X_c,
			var_joint_axis,
			var_joint_q_start,
			var_joint_qd_start,
			var_joint_q,
			var_joint_qd,
			adj_body_q,
			adj_body_qd,
			adj_body_com,
			adj_joint_type,
			adj_joint_parent,
			adj_joint_X_p,
			adj_joint_X_c,
			adj_joint_axis,
			adj_joint_q_start,
			adj_joint_qd_start,
			adj_joint_q,
			adj_joint_qd);
}

} // extern C

